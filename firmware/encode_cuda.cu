#include "hip/hip_runtime.h"
#include "encode_cuda.cuh"
#include <opencv2/core.hpp> 
#include <opencv2/imgcodecs.hpp>
#include <>
#include <hip/device_functions.h>
#include <hip/hip_texture_types.h>
#include <texture_types.h>
#include "hip/hip_runtime.h" 
#include <hip/hip_runtime.h>

#include <iostream>
#include <stdint.h>
#include <vector>  
#include "easylogging++.h"
#include "protocol.h"


int patterns_count_ = 36;
int wrap_count_ = 8;
int unwrap_count_ = 2;
int image_width_ = 1920;
int image_height_ = 1200;

__device__ int d_image_width_ = 1920;
__device__ int d_image_height_ = 1200;
__device__ float d_confidence_ = 1200;


unsigned char* d_patterns_list_hdr_0[36];
float* d_confidence_list_hdr_0[8];
float* d_wrap_map_list_hdr_0[8];
float* d_unwrap_map_list_hdr_0[2];

unsigned char* d_patterns_list_hdr_1[36];
float* d_confidence_list_hdr_1[8];
float* d_wrap_map_list_hdr_1[8];
float* d_unwrap_map_list_hdr_1[2];

unsigned char* d_patterns_list_hdr_2[36];
float* d_confidence_list_hdr_2[8];
float* d_wrap_map_list_hdr_2[8];
float* d_unwrap_map_list_hdr_2[2];

#define D_HDR_MAX_NUM 6

float* d_hdr_depth_map_list_[D_HDR_MAX_NUM];
unsigned char* d_hdr_brightness_list_[D_HDR_MAX_NUM];
float* d_hdr_bright_pixel_sum_list_[D_HDR_MAX_NUM];
float* d_hdr_depth_map_;
unsigned char* d_hdr_brightness_;

#define D_REPETITIONB_MAX_NUM 10
unsigned char* d_repetition_patterns_list_[6*D_REPETITIONB_MAX_NUM]; 
unsigned short* d_repetition_merge_patterns_list_[6];  

#define D_REPETITION_02_MAX_NUM 37
__device__ unsigned short* d_repetition_02_merge_patterns_list_[D_REPETITION_02_MAX_NUM];  
/*****************************************************************************/

texture<unsigned char, 1, hipReadModeElementType> texture_patterns_0;
texture<unsigned char, 1, hipReadModeElementType> texture_patterns_1;
texture<unsigned char, 1, hipReadModeElementType> texture_patterns_2;
texture<unsigned char, 1, hipReadModeElementType> texture_patterns_3;
texture<unsigned char, 1, hipReadModeElementType> texture_patterns_4;
texture<unsigned char, 1, hipReadModeElementType> texture_patterns_5;
texture<unsigned char, 1, hipReadModeElementType> texture_patterns_6;
texture<unsigned char, 1, hipReadModeElementType> texture_patterns_7;
texture<unsigned char, 1, hipReadModeElementType> texture_patterns_8;
texture<unsigned char, 1, hipReadModeElementType> texture_patterns_9;
texture<unsigned char, 1, hipReadModeElementType> texture_patterns_10;
texture<unsigned char, 1, hipReadModeElementType> texture_patterns_11;
texture<unsigned char, 1, hipReadModeElementType> texture_patterns_12;
texture<unsigned char, 1, hipReadModeElementType> texture_patterns_13;
texture<unsigned char, 1, hipReadModeElementType> texture_patterns_14;
texture<unsigned char, 1, hipReadModeElementType> texture_patterns_15;
texture<unsigned char, 1, hipReadModeElementType> texture_patterns_16;
texture<unsigned char, 1, hipReadModeElementType> texture_patterns_17;
texture<unsigned char, 1, hipReadModeElementType> texture_patterns_18;

__device__ unsigned char* d_patterns_list[36];
__device__ unsigned char* d_brightness_;
__device__ float* d_confidence_list[8];
__device__ float* d_wrap_map_list[8];
__device__ float* d_unwrap_map_list[2];
__device__ float* d_point_cloud_map_;
__device__ float* d_depth_map_;
__device__ short* d_depth_map_short_;
__device__ float* d_triangulation_error_map_;

unsigned char* d_mask_;
/*********************************************************************************/
float* d_camera_intrinsic_;
float* d_project_intrinsic_;
float* d_camera_distortion_;
float* d_projector_distortion_;
float* d_rotation_matrix_;
float* d_translation_matrix_;

__device__ int d_dlp_width_ = 1920;
__device__ int d_dlp_height_ = 1080;
__device__ float d_max_phase_ = 2* 3.1415926535;

bool load_calib_data_flag_ = false;

#define DF_PI 3.1415926535
/*********************************************************************************/


__device__ float d_baseline_ = 0;
 
// 因为有多个查找表，在上传查找表前，先释放内存，防止内存泄漏
__device__ float* d_single_pattern_mapping_ = NULL;
__device__ float* d_single_pattern_minimapping_ = NULL;
__device__ float* d_xL_rotate_x_ = NULL;
__device__ float* d_xL_rotate_y_ = NULL; 
__device__ float* d_R_1_ = NULL; 

  
/*********************************************************************************************/


dim3 threadsPerBlock(8, 8);
dim3 blocksPerGrid((image_width_ + threadsPerBlock.x - 1) / threadsPerBlock.x,
(image_height_ + threadsPerBlock.y - 1) / threadsPerBlock.y);


#define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}


void cuda_set_config(struct SystemConfigDataStruct param)
{ 
	hipMemcpyToSymbol(HIP_SYMBOL(d_confidence_), &param.Instance().firwmare_param_.confidence, sizeof(float));
}

bool cuda_set_camera_version(int version)
{
    switch (version)
    {
    case DFX_800:
    {
		int dlp_width = 1280;
		int dlp_height = 720;
		hipMemcpyToSymbol(HIP_SYMBOL(d_dlp_width_), &dlp_width, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(d_dlp_height_), &dlp_height, sizeof(int));
  
		int camera_width = 1920;
		int camera_height = 1200;
		hipMemcpyToSymbol(HIP_SYMBOL(d_image_width_), &camera_width, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(d_image_height_), &camera_height, sizeof(int));

        return true;
    }
    break;

    case DFX_1800:
    {
		int dlp_width = 1920;
		int dlp_height = 1080;
		hipMemcpyToSymbol(HIP_SYMBOL(d_dlp_width_), &dlp_width, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(d_dlp_height_), &dlp_height, sizeof(int));

		int camera_width = 1920;
		int camera_height = 1200;
		hipMemcpyToSymbol(HIP_SYMBOL(d_image_width_), &camera_width, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(d_image_height_), &camera_height, sizeof(int));
        return true;
    }
    break;

    default:
        break;
    }

	return false;
}


/***********************************************************************************************************************************************/

bool parallel_cuda_copy_signal_patterns(unsigned char* patterns_ptr,int serial_flag)
{
	CHECK(hipMemcpyAsync(d_patterns_list[serial_flag], patterns_ptr, image_height_*image_width_ * sizeof(unsigned char), hipMemcpyHostToDevice)); 
}

bool parallel_cuda_copy_repetition_signal_patterns(unsigned char* patterns_ptr,int serial_flag)
{
	CHECK(hipMemcpyAsync(d_repetition_patterns_list_[serial_flag], patterns_ptr, image_height_*image_width_ * sizeof(unsigned char), hipMemcpyHostToDevice));
}


bool parallel_cuda_clear_repetition_02_patterns()
{
	for(int i = 0;i< D_REPETITION_02_MAX_NUM;i++)
	{ 
		 hipMemset(d_repetition_02_merge_patterns_list_[i], 0,image_width_* image_height_*sizeof(ushort));
		// CHECK(hipMemcpyAsync(d_repetition_02_merge_patterns_list_[i], &val,image_width_* image_height_*sizeof(ushort), hipMemcpyHostToDevice));
	}
	// hipDeviceSynchronize();
  
  return true;
}

bool parallel_cuda_merge_repetition_02_patterns(int repetition_serial)
{
	// int merge_serial = repetition_serial%19; 
	cuda_merge_pattern<< <blocksPerGrid, threadsPerBlock >> >(d_patterns_list[repetition_serial],image_height_, image_width_,d_repetition_02_merge_patterns_list_[repetition_serial]);

	return true;
}

bool parallel_cuda_merge_repetition_patterns(int repetition_serial)
{

	int merge_serial = repetition_serial%6; 
	cuda_merge_pattern<< <blocksPerGrid, threadsPerBlock >> >(d_repetition_patterns_list_[repetition_serial],image_height_, image_width_,d_repetition_merge_patterns_list_[merge_serial]);

	return true;
}

__global__ void cuda_merge_pattern(unsigned char * const d_in_pattern,uint32_t img_height, uint32_t img_width,unsigned short * const d_out_merge_pattern)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * img_width + idx;

	if (idx < img_width && idy < img_height)
	{
  
		d_out_merge_pattern[offset] += d_in_pattern[offset];  

	}
}


bool parallel_cuda_compute_model_02_merge_repetition_02_phase(int repetition_count)
{
	int i = 0;
	cuda_merge_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[i+ 0], d_repetition_02_merge_patterns_list_[i+ 1],
		d_repetition_02_merge_patterns_list_[i+ 2],d_repetition_02_merge_patterns_list_[i+ 3],repetition_count, d_wrap_map_list[0], d_confidence_list[0]);
			
	i = 4;
	cuda_merge_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[i+ 0], d_repetition_02_merge_patterns_list_[i+ 1],
		d_repetition_02_merge_patterns_list_[i+ 2],d_repetition_02_merge_patterns_list_[i+ 3],repetition_count, d_wrap_map_list[1], d_confidence_list[1]);

	i = 8;
	cuda_merge_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[i+ 0], d_repetition_02_merge_patterns_list_[i+ 1],
		d_repetition_02_merge_patterns_list_[i+ 2],d_repetition_02_merge_patterns_list_[i+ 3],repetition_count, d_wrap_map_list[2], d_confidence_list[2]);
	
	i = 12;
	cuda_merge_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[i+ 0], d_repetition_02_merge_patterns_list_[i+ 1],
		d_repetition_02_merge_patterns_list_[i+ 2],d_repetition_02_merge_patterns_list_[i+ 3],d_repetition_02_merge_patterns_list_[i+ 4],d_repetition_02_merge_patterns_list_[i+ 5] ,
		repetition_count,image_height_, image_width_, d_wrap_map_list[3], d_confidence_list[3]);

	i = 18;
	cuda_merge_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[i+ 0], d_repetition_02_merge_patterns_list_[i+ 1],
		d_repetition_02_merge_patterns_list_[i+ 2],d_repetition_02_merge_patterns_list_[i+ 3],repetition_count, d_wrap_map_list[4], d_confidence_list[4]);
			
	i = 22;
	cuda_merge_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[i+ 0], d_repetition_02_merge_patterns_list_[i+ 1],
		d_repetition_02_merge_patterns_list_[i+ 2],d_repetition_02_merge_patterns_list_[i+ 3],repetition_count, d_wrap_map_list[5], d_confidence_list[5]);

	i = 26;
	cuda_merge_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[i+ 0], d_repetition_02_merge_patterns_list_[i+ 1],
		d_repetition_02_merge_patterns_list_[i+ 2],d_repetition_02_merge_patterns_list_[i+ 3],repetition_count, d_wrap_map_list[6], d_confidence_list[6]);
	
	i = 30;
	cuda_merge_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[i+ 0], d_repetition_02_merge_patterns_list_[i+ 1],
		d_repetition_02_merge_patterns_list_[i+ 2],d_repetition_02_merge_patterns_list_[i+ 3],d_repetition_02_merge_patterns_list_[i+ 4],d_repetition_02_merge_patterns_list_[i+ 5] ,
		repetition_count,image_height_, image_width_, d_wrap_map_list[7], d_confidence_list[7]);

	cuda_merge_brigntness_map<< <blocksPerGrid, threadsPerBlock >> >(d_repetition_02_merge_patterns_list_[36],repetition_count,d_brightness_);
}

bool parallel_cuda_compute_merge_repetition_02_phase(int repetition_count)
{
	
	cuda_merge_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[0], d_repetition_02_merge_patterns_list_[1],
		d_repetition_02_merge_patterns_list_[2],d_repetition_02_merge_patterns_list_[3],repetition_count, d_wrap_map_list[0], d_confidence_list[0]);
			
	cuda_merge_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[4], d_repetition_02_merge_patterns_list_[5],
		d_repetition_02_merge_patterns_list_[6],d_repetition_02_merge_patterns_list_[7],repetition_count, d_wrap_map_list[1], d_confidence_list[1]);

	cuda_merge_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[8], d_repetition_02_merge_patterns_list_[9],
		d_repetition_02_merge_patterns_list_[10],d_repetition_02_merge_patterns_list_[11],repetition_count, d_wrap_map_list[2], d_confidence_list[2]);
	
	cuda_merge_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_02_merge_patterns_list_[12], d_repetition_02_merge_patterns_list_[13],
		d_repetition_02_merge_patterns_list_[14],d_repetition_02_merge_patterns_list_[15],d_repetition_02_merge_patterns_list_[16],d_repetition_02_merge_patterns_list_[17] ,
		repetition_count,image_height_, image_width_, d_wrap_map_list[3], d_confidence_list[3]);

	return true;
}

bool parallel_cuda_compute_merge_phase(int repetition_count)
{

	cuda_merge_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_repetition_merge_patterns_list_[0], d_repetition_merge_patterns_list_[1],
		d_repetition_merge_patterns_list_[2],d_repetition_merge_patterns_list_[3],d_repetition_merge_patterns_list_[4],d_repetition_merge_patterns_list_[5] ,
		repetition_count,image_height_, image_width_, d_wrap_map_list[3], d_confidence_list[3]);

	return true;
}

__global__ void cuda_merge_six_step_phase_shift(unsigned short * const d_in_0, unsigned short * const d_in_1, unsigned short * const d_in_2, 
	unsigned short * const d_in_3,unsigned short* const d_in_4,unsigned short* const d_in_5,int repetition_count,
	uint32_t img_height, uint32_t img_width,float * const d_out, float * const confidence)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * img_width + idx;
	float s_0 =  0;
	float s_1 =  0.866025;
	float s_2 =  0.866025;
	float s_3 =  0;
	float s_4 =  -0.866025;
	float s_5 =  -0.866025;
	float c_0 =  1;
	float c_1 =  0.5;
	float c_2 =  -0.5;
	float c_3 =  -1;
	float c_4 =  -0.5;
	float c_5 =  0.5;
	
	if (idx < img_width && idy < img_height)
	{

		float a = c_0 *d_in_3[offset] + c_1 *d_in_4[offset] + c_2 *d_in_5[offset] + c_3* d_in_0[offset] +c_4*d_in_1[offset] + c_5*d_in_2[offset];
		float b = s_0 *d_in_3[offset] + s_1 *d_in_4[offset] + s_2 *d_in_5[offset] + s_3* d_in_0[offset] +s_4*d_in_1[offset] + s_5*d_in_2[offset];

  
		confidence[offset] = std::sqrt(a*a + b*b);
		d_out[offset] = DF_PI + std::atan2(a, b);
	}

	
}

/***********************************************************************************************************************************************************/

bool parallel_cuda_compute_phase(int serial_flag)
{
	 
	switch(serial_flag)
	{
		case 0:
		{ 
			// int i= 0;
			// cuda_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list[i+0], d_patterns_list[i + 1], d_patterns_list[i + 2],
			// 	d_patterns_list[i + 3], image_height_, image_width_, d_wrap_map_list[serial_flag], d_confidence_list[serial_flag]);

				cuda_four_step_phase_shift_texture<< <blocksPerGrid, threadsPerBlock >> >(serial_flag,d_wrap_map_list[serial_flag], d_confidence_list[serial_flag]);
		}
		break;
		case 1:
		{

			// int i= 4;
			// cuda_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list[i+0], d_patterns_list[i + 1], d_patterns_list[i + 2],
			// 	d_patterns_list[i + 3], image_height_, image_width_, d_wrap_map_list[serial_flag], d_confidence_list[serial_flag]);
				
				cuda_four_step_phase_shift_texture<< <blocksPerGrid, threadsPerBlock >> >(serial_flag,d_wrap_map_list[serial_flag], d_confidence_list[serial_flag]);
			
		}
		break;
		case 2:
		{ 
			// int i= 8;
			// cuda_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list[i+0], d_patterns_list[i + 1], d_patterns_list[i + 2],
			// 	d_patterns_list[i + 3], image_height_, image_width_, d_wrap_map_list[serial_flag], d_confidence_list[serial_flag]);
				
				cuda_four_step_phase_shift_texture<< <blocksPerGrid, threadsPerBlock >> >(serial_flag,d_wrap_map_list[serial_flag], d_confidence_list[serial_flag]);
		}
		break;
		case 3:
		{ 
			// int i= 12; 
			// cuda_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list[i+0], d_patterns_list[i + 1], d_patterns_list[i + 2],
			// 	d_patterns_list[i + 3],d_patterns_list[i + 4],d_patterns_list[i + 5] ,
			// 	 image_height_, image_width_, d_wrap_map_list[serial_flag], d_confidence_list[serial_flag]);
 
				cuda_six_step_phase_shift_texture<< <blocksPerGrid, threadsPerBlock >> > (d_wrap_map_list[serial_flag], d_confidence_list[serial_flag]);
				// hipDeviceSynchronize();

				// cv::Mat phase(1200, 1920, CV_32F, cv::Scalar(0));
				// CHECK(hipMemcpy(phase.data, d_wrap_map_list[serial_flag], 1 * image_height_ * image_width_ * sizeof(float), hipMemcpyDeviceToHost));
				// cv::imwrite("phase1.tiff",phase);
		}
		break;
		case 4:
		{
			int i= 18;
			cuda_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list[i+0], d_patterns_list[i + 1], d_patterns_list[i + 2],
				d_patterns_list[i + 3], image_height_, image_width_, d_wrap_map_list[serial_flag], d_confidence_list[serial_flag]);
		}
		break;
		case 5:
		{
			int i= 22;
			cuda_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list[i+0], d_patterns_list[i + 1], d_patterns_list[i + 2],
				d_patterns_list[i + 3], image_height_, image_width_, d_wrap_map_list[serial_flag], d_confidence_list[serial_flag]);
		}
		break;
		case 6:
		{
			int i= 26;
			cuda_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list[i+0], d_patterns_list[i + 1], d_patterns_list[i + 2],
				d_patterns_list[i + 3], image_height_, image_width_, d_wrap_map_list[serial_flag], d_confidence_list[serial_flag]);
		}
		break;
  
		default :
			break;
	}

	
	
	return true;
}


bool parallel_cuda_unwrap_phase(int serial_flag)
{

	switch(serial_flag)
	{
		case 0:
		{ 

		}
		break;
		case 1:
		{ 
			// CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuda_variable_phase_unwrap), hipFuncCachePreferL1) );
            cuda_variable_phase_unwrap<< <blocksPerGrid, threadsPerBlock >> >(d_wrap_map_list[0], d_wrap_map_list[1], 8.0,
				image_height_, image_width_,CV_PI, d_unwrap_map_list[0]);
            // CHECK ( hipGetLastError () );

			// cuda_variable_phase_unwrap << <blocksPerGrid, threadsPerBlock >> >(d_wrap_map_list[0], d_wrap_map_list[1], 8.0,
			// 	image_height_, image_width_, d_unwrap_map_list[0]);
				
			
		}
		break;
		case 2:
		{ 
			// CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuda_variable_phase_unwrap), hipFuncCachePreferL1) );
			cuda_variable_phase_unwrap << <blocksPerGrid, threadsPerBlock >> >(d_unwrap_map_list[0], d_wrap_map_list[2], 4.0,
				image_height_, image_width_,CV_PI, d_unwrap_map_list[0]); 
			// CHECK ( hipGetLastError () );
		}
		break;
		case 3:
		{ 
			// CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuda_variable_phase_unwrap), hipFuncCachePreferL1) );
			cuda_variable_phase_unwrap << <blocksPerGrid, threadsPerBlock >> >(d_unwrap_map_list[0], d_wrap_map_list[3], 4.0,
				image_height_, image_width_,1.5, d_unwrap_map_list[0]); 

			// CHECK ( hipGetLastError () );
		}
		break;
		case 4:
		{
 
		}
		break;
		case 5:
		{
			cuda_variable_phase_unwrap << <blocksPerGrid, threadsPerBlock >> >(d_wrap_map_list[4], d_wrap_map_list[5], 8.0,
				image_height_, image_width_,CV_PI, d_unwrap_map_list[1]);
		}
		break;
		case 6:
		{
			cuda_variable_phase_unwrap << <blocksPerGrid, threadsPerBlock >> >(d_unwrap_map_list[1], d_wrap_map_list[6], 4.0,
				image_height_, image_width_,CV_PI, d_unwrap_map_list[1]);
			// cuda_normalize_phase << <blocksPerGrid, threadsPerBlock >> >(d_unwrap_map_list[0],128.0, d_unwrap_map_list[1],18.0,
			// image_height_, image_width_, d_unwrap_map_list[0],d_unwrap_map_list[1]);
			
			LOG(INFO)<<"unwrap 6:  ";

		}
		break;
		case 7:
		{
			cuda_variable_phase_unwrap << <blocksPerGrid, threadsPerBlock >> >(d_unwrap_map_list[1], d_wrap_map_list[7], 4.0,
				image_height_, image_width_,CV_PI, d_unwrap_map_list[1]);
			cuda_normalize_phase << <blocksPerGrid, threadsPerBlock >> >(d_unwrap_map_list[0],128.0, d_unwrap_map_list[1],72.0,
			image_height_, image_width_, d_unwrap_map_list[0],d_unwrap_map_list[1]);
			
			LOG(INFO)<<"unwrap 7:  ";

		}
		break;
 

		default :
			break;
	}


	return true;
}


bool parallel_cuda_reconstruct()
{
	cuda_rebuild << <blocksPerGrid, threadsPerBlock >> >(d_unwrap_map_list[0], d_unwrap_map_list[1],d_camera_intrinsic_,d_camera_distortion_,
		d_project_intrinsic_,d_projector_distortion_,d_rotation_matrix_,d_translation_matrix_, 
		d_point_cloud_map_,d_depth_map_, d_triangulation_error_map_, d_confidence_list[3]);

  
	hipDeviceSynchronize();

	
    LOG(INFO)<<"rebuild data!";
	 

    // LOG(INFO)<<"unwrap_0";

	// cv::Mat confidence_map(image_height_, image_width_, CV_32F, cv::Scalar(0));
	// hipMemcpy(confidence_map.data, d_confidence_list[3], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);

	// cv::Mat wrap_1(image_height_, image_width_, CV_32F, cv::Scalar(0));
	// hipMemcpy(wrap_1.data, d_wrap_map_list[1], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);
	// cv::Mat wrap_2(image_height_, image_width_, CV_32F, cv::Scalar(0));
	// hipMemcpy(wrap_2.data, d_wrap_map_list[2], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);

	// cv::Mat wrap_3(image_height_, image_width_, CV_32F, cv::Scalar(0));
	// hipMemcpy(wrap_3.data, d_wrap_map_list[3], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);

	// cv::imwrite("confidence_map.tiff",confidence_map);
	// cv::imwrite("wrap_3.tiff",wrap_3);
	// cv::imwrite("wrap_1.tiff",wrap_1);
	// cv::imwrite("wrap_2.tiff",wrap_2);

	// cv::Mat unwrap_0(image_height_, image_width_, CV_32F, cv::Scalar(0));
	// hipMemcpy(unwrap_0.data, d_unwrap_map_list[0], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);

	// cv::Mat unwrap_1(image_height_, image_width_, CV_32F, cv::Scalar(0));
	// hipMemcpy(unwrap_1.data,d_unwrap_map_list[1], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);

	// cv::Mat deep_map(image_height_, image_width_, CV_32F, cv::Scalar(0));
	// hipMemcpy(deep_map.data, d_depth_map_, image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);

	// cv::Mat err_map(image_height_, image_width_, CV_32F, cv::Scalar(0));
	// hipMemcpy(err_map.data, d_triangulation_error_map_, image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);

	// cv::Mat points_map(image_height_, image_width_, CV_32FC3, cv::Scalar(0));
	// hipMemcpy(points_map.data, d_point_cloud_map_, 3*image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);

 
	
    // LOG(INFO)<<"copy data!";

	// cv::imwrite("unwrap_map_0.tiff",unwrap_0);
	// cv::imwrite("unwrap_map_1.tiff",unwrap_1);
	// cv::imwrite("deep_map.tiff",deep_map);
	// cv::imwrite("err_map.tiff",err_map);
	// cv::imwrite("points_map.tiff",points_map);
	
    // LOG(INFO)<<"rebuild data!";


}


bool parallel_cuda_copy_pointcloud_from_gpu(float* pointcloud,unsigned char* brightness)
{
	if(!load_calib_data_flag_)
	{
		return false;
	} 

	LOG(INFO)<<"copy......"; 
	cuda_get_brightness_data(brightness); 
	cuda_get_pointcloud_data(pointcloud); 
	LOG(INFO)<<"copy result";
}

bool parallel_cuda_copy_result_from_gpu(float* depth,unsigned char* brightness)
{
	if(!load_calib_data_flag_)
	{
		return false;
	} 

	LOG(INFO)<<"copy......"; 
	cuda_get_brightness_data(brightness); 
	cuda_get_depth_data(depth); 
	LOG(INFO)<<"copy result";
 
}

bool parallel_cuda_copy_unwrap_phase_from_gpu(int serial_flag,float* unwrap_map)
{

	CHECK(hipMemcpy(unwrap_map, d_unwrap_map_list[serial_flag], 1 * image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost));
	return true;
}

bool parallel_cuda_copy_unwrap_phase_to_gpu(int serial_flag,float* unwrap_map)
{

	CHECK(hipMemcpy(d_unwrap_map_list[serial_flag], unwrap_map,  1 * image_height_*image_width_ * sizeof(float), hipMemcpyHostToDevice));
	return true;
}

void BubbleSort(float  *p, int length, int * ind_diff)
{
	for (int m = 0; m < length; m++)
	{
		ind_diff[m] = m;
	}
 
	for (int i = 0; i < length; i++)
	{
		for (int j = 0; j < length- i - 1; j++)
		{
			if (p[j] > p[j + 1])
			{
				float temp = p[j];
				p[j] = p[j + 1];
				p[j + 1] = temp;
 
				int ind_temp = ind_diff[j];
				ind_diff[j] = ind_diff[j + 1];
				ind_diff[j + 1] = ind_temp;
			}
		}
	}
}

 

bool parallel_cuda_merge_hdr_data(int hdr_num,float* depth_map, unsigned char* brightness)
{
	
	LOG(INFO)<<"sum pixels ";
	float sum_pixels_list[6];  

    for(int i= 0;i<hdr_num;i++)
    { 
		CHECK(hipMemcpy(&sum_pixels_list[i], d_hdr_bright_pixel_sum_list_[i], 1* sizeof(float), hipMemcpyDeviceToHost));
    }
 
 
	std::vector<float> param_list;
	std::vector<int> id; 
	std::vector<bool> flag_list;

	for (int i = 0; i < hdr_num; i++)
	{ 
        param_list.push_back(sum_pixels_list[i]);
		id.push_back(0);
		flag_list.push_back(true);
    } 
   	std::sort(param_list.begin(),param_list.end(),std::greater<float>());
 
 
	for (int i = 0; i < hdr_num; i++)
	{ 
		
		for(int j= 0;j< hdr_num;j++)
		{
			if(param_list[i] == sum_pixels_list[j])
			{
				if(flag_list[j])
				{ 
					id[i] = j;
					flag_list[j] = false; 
					break;
				}
			}
		}
		 
    } 

 
	for (int i = 0; i < hdr_num; i++)
	{ 
        LOG(INFO)<<"sum pixels "<<i<<": "<<sum_pixels_list[i]<<" _ "<<id[i];
    }
 

	switch(hdr_num)
	{
		case 1:
		{

			CHECK(hipMemcpy(depth_map, d_hdr_depth_map_list_[0], 1 * image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_list_[0], 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));
		} 
		break;
		case 2:
		{
			parallel_cuda_merge_hdr_2 << <blocksPerGrid, threadsPerBlock >> > (d_hdr_depth_map_list_[id[0]],d_hdr_depth_map_list_[id[1]], d_hdr_brightness_list_[id[0]], 
				d_hdr_brightness_list_[id[1]], image_height_, image_width_, d_hdr_depth_map_,d_hdr_brightness_);

				
			CHECK(hipMemcpy(depth_map, d_hdr_depth_map_, 1 * image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));

		}
		break;
		case 3:
		{
			parallel_cuda_merge_hdr_3 << <blocksPerGrid, threadsPerBlock >> > (d_hdr_depth_map_list_[id[0]],d_hdr_depth_map_list_[id[1]],d_hdr_depth_map_list_[id[2]], d_hdr_brightness_list_[id[0]], 
				d_hdr_brightness_list_[id[1]], d_hdr_brightness_list_[id[2]], image_height_, image_width_, d_hdr_depth_map_,d_hdr_brightness_);
				
			CHECK(hipMemcpy(depth_map, d_hdr_depth_map_, 1 * image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));

		}
		break;
		case 4:
		{
			parallel_cuda_merge_hdr_4 << <blocksPerGrid, threadsPerBlock >> > (d_hdr_depth_map_list_[id[0]],d_hdr_depth_map_list_[id[1]],d_hdr_depth_map_list_[id[2]],d_hdr_depth_map_list_[id[3]],
				 d_hdr_brightness_list_[id[0]], d_hdr_brightness_list_[id[1]], d_hdr_brightness_list_[id[2]], d_hdr_brightness_list_[id[3]], 
				image_height_, image_width_, d_hdr_depth_map_,d_hdr_brightness_);
				
			CHECK(hipMemcpy(depth_map, d_hdr_depth_map_, 1 * image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));

		}
		break;
		case 5:
		{
			parallel_cuda_merge_hdr_5 << <blocksPerGrid, threadsPerBlock >> > (d_hdr_depth_map_list_[id[0]],d_hdr_depth_map_list_[id[1]],d_hdr_depth_map_list_[id[2]],
				d_hdr_depth_map_list_[id[3]],d_hdr_depth_map_list_[id[4]],
				 d_hdr_brightness_list_[id[0]], d_hdr_brightness_list_[id[1]], d_hdr_brightness_list_[id[2]], d_hdr_brightness_list_[id[3]], d_hdr_brightness_list_[id[4]], 
				image_height_, image_width_, d_hdr_depth_map_,d_hdr_brightness_);
				
			CHECK(hipMemcpy(depth_map, d_hdr_depth_map_, 1 * image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));

		}
		break;
		case 6:
		{
			parallel_cuda_merge_hdr_6 << <blocksPerGrid, threadsPerBlock >> > (d_hdr_depth_map_list_[id[0]],d_hdr_depth_map_list_[id[1]],d_hdr_depth_map_list_[id[2]],
				d_hdr_depth_map_list_[id[3]],d_hdr_depth_map_list_[id[4]],d_hdr_depth_map_list_[id[5]],
				 d_hdr_brightness_list_[id[0]], d_hdr_brightness_list_[id[1]], d_hdr_brightness_list_[id[2]], d_hdr_brightness_list_[id[3]], d_hdr_brightness_list_[id[4]], 
				 d_hdr_brightness_list_[id[5]], 
				image_height_, image_width_, d_hdr_depth_map_,d_hdr_brightness_);
				
			CHECK(hipMemcpy(depth_map, d_hdr_depth_map_, 1 * image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost));
			// CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));

		}
		break;

		default:
		 		return false;

	}

 	// CHECK(hipMemcpy(brightness, d_hdr_brightness_list_[id[0]], 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));
 	CHECK(hipMemcpy(brightness, d_hdr_brightness_list_[hdr_num-1], 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));
	LOG(INFO)<<"DHR Finished!";

	return true;
}

__global__ void parallel_cuda_merge_hdr_6(const float*  depth_map_0,const float*  depth_map_1,const float*  depth_map_2,
	const float*  depth_map_3,const float*  depth_map_4,const float*  depth_map_5,
	const unsigned char* brightness_0,const unsigned char* brightness_1,const unsigned char* brightness_2,
	const unsigned char* brightness_3,const unsigned char* brightness_4,const unsigned char* brightness_5,
	uint32_t img_height, uint32_t img_width, float* const depth_map,unsigned char * const brightness)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * img_width + idx;

	if (idx < img_width && idy < img_height)
	{
 

		float pixel= 0;
		pixel +=  brightness_0[offset];
		pixel +=  brightness_1[offset];
		pixel +=  brightness_2[offset];
		pixel +=  brightness_3[offset];
		pixel +=  brightness_4[offset];
		pixel +=  brightness_5[offset];

		pixel/= 6.0;


		brightness[offset] = pixel;

		if(brightness_0[offset] < 255)
		{
			// brightness[offset] = brightness_0[offset];
			depth_map[offset] = depth_map_0[offset];
		}

		else if(brightness_1[offset] < 255)
		{
			// brightness[offset] = brightness_1[offset];
			depth_map[offset] = depth_map_1[offset];
		}
		else if(brightness_2[offset] < 255)
		{
			// brightness[offset] = brightness_1[offset];
			depth_map[offset] = depth_map_2[offset];
		}
		else if(brightness_3[offset] < 255)
		{
			// brightness[offset] = brightness_1[offset];
			depth_map[offset] = depth_map_3[offset];
		}
		else if(brightness_4[offset] < 255)
		{
			// brightness[offset] = brightness_1[offset];
			depth_map[offset] = depth_map_4[offset];
		}
		else
		{	
			// brightness[offset] = brightness_2[offset];
			depth_map[offset] = depth_map_5[offset];
		}
		//没有深度则用最亮的深度值
		if (depth_map[offset] <= 0)
		{
			depth_map[offset] = depth_map_0[offset];
		}
	}
}

__global__ void parallel_cuda_merge_hdr_5(const float*  depth_map_0,const float*  depth_map_1,const float*  depth_map_2,
	const float*  depth_map_3,const float*  depth_map_4,
	const unsigned char* brightness_0,const unsigned char* brightness_1,const unsigned char* brightness_2,
	const unsigned char* brightness_3,const unsigned char* brightness_4,
	uint32_t img_height, uint32_t img_width, float* const depth_map,unsigned char * const brightness)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * img_width + idx;

	if (idx < img_width && idy < img_height)
	{



		float pixel= 0;
		pixel +=  brightness_0[offset];
		pixel +=  brightness_1[offset];
		pixel +=  brightness_2[offset];
		pixel +=  brightness_3[offset];
		pixel +=  brightness_4[offset];

		pixel/= 5.0;


		brightness[offset] = pixel;

		if(brightness_0[offset] < 255)
		{
			// brightness[offset] = brightness_0[offset];
			depth_map[offset] = depth_map_0[offset];
		}

		else if(brightness_1[offset] < 255)
		{
			// brightness[offset] = brightness_1[offset];
			depth_map[offset] = depth_map_1[offset];
		}
		else if(brightness_2[offset] < 255)
		{
			// brightness[offset] = brightness_1[offset];
			depth_map[offset] = depth_map_2[offset];
		}
		else if(brightness_3[offset] < 255)
		{
			// brightness[offset] = brightness_1[offset];
			depth_map[offset] = depth_map_3[offset];
		}
		else
		{	
			// brightness[offset] = brightness_2[offset];
			depth_map[offset] = depth_map_4[offset];
		}
		//没有深度则用最亮的深度值
		if (depth_map[offset] <= 0)
		{
			depth_map[offset] = depth_map_0[offset];
		}
	}
}


__global__ void parallel_cuda_merge_hdr_4(const float*  depth_map_0,const float*  depth_map_1,const float*  depth_map_2,const float*  depth_map_3,
	const unsigned char* brightness_0,const unsigned char* brightness_1,const unsigned char* brightness_2,const unsigned char* brightness_3,
	uint32_t img_height, uint32_t img_width, float* const depth_map,unsigned char * const brightness)
	{
		const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
		const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
		const unsigned int offset = idy * img_width + idx;
	
		if (idx < img_width && idy < img_height)
		{
	
	
	
			float pixel= 0;
			pixel +=  brightness_0[offset];
			pixel +=  brightness_1[offset];
			pixel +=  brightness_2[offset];
			pixel +=  brightness_3[offset];
	
			pixel/= 4.0;
	
	
			brightness[offset] = pixel;
	
			if(brightness_0[offset] < 255)
			{
				// brightness[offset] = brightness_0[offset];
				depth_map[offset] = depth_map_0[offset];
			}
	
			else if(brightness_1[offset] < 255)
			{
				// brightness[offset] = brightness_1[offset];
				depth_map[offset] = depth_map_1[offset];
			}
			else if(brightness_2[offset] < 255)
			{
				// brightness[offset] = brightness_1[offset];
				depth_map[offset] = depth_map_2[offset];
			}
			else
			{	
				// brightness[offset] = brightness_2[offset];
				depth_map[offset] = depth_map_3[offset];
			}
			//没有深度则用最亮的深度值
			if (depth_map[offset] <= 0)
			{
				depth_map[offset] = depth_map_0[offset];
			}
		}
	}

__global__ void parallel_cuda_merge_hdr_3(const float*  depth_map_0,const float*  depth_map_1,const float*  depth_map_2,const unsigned char* brightness_0,const unsigned char* brightness_1,
	const unsigned char* brightness_2,uint32_t img_height, uint32_t img_width, float* const depth_map,unsigned char * const brightness)
	{
		const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
		const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
		const unsigned int offset = idy * img_width + idx;
	
		if (idx < img_width && idy < img_height)
		{
	
	
	
			float pixel= 0;
			pixel +=  brightness_0[offset];
			pixel +=  brightness_1[offset];
			pixel +=  brightness_2[offset];
	
			pixel/= 3.0;
	
	
			brightness[offset] = pixel;
	
			if(brightness_0[offset] < 255)
			{
				// brightness[offset] = brightness_0[offset];
				depth_map[offset] = depth_map_0[offset];
			}
	
			else if(brightness_1[offset] < 255)
			{
				// brightness[offset] = brightness_1[offset];
				depth_map[offset] = depth_map_1[offset];
			}
			else
			{	
				// brightness[offset] = brightness_2[offset];
				depth_map[offset] = depth_map_2[offset];
			}
				//没有深度则用最亮的深度值
			if(depth_map[offset]<= 0)
			{
				depth_map[offset] = depth_map_0[offset];
			}
	
		}
	}

__global__ void parallel_cuda_merge_hdr_2(const float*  depth_map_0,const float*  depth_map_1,const unsigned char* brightness_0,const unsigned char* brightness_1,
	uint32_t img_height, uint32_t img_width, float* const depth_map,unsigned char * const brightness)
	{
		const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
		const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
		const unsigned int offset = idy * img_width + idx;
	
		if (idx < img_width && idy < img_height)
		{
	
			float pixel= 0;
			pixel +=  brightness_0[offset];
			pixel +=  brightness_1[offset];
	
			pixel/= 2.0;
	
	
			brightness[offset] = pixel;
	
			if(brightness_0[offset] < 255)
			{
				// brightness[offset] = brightness_0[offset];
				depth_map[offset] = depth_map_0[offset];
			}
			else 
			{
				// brightness[offset] = brightness_1[offset];
				depth_map[offset] = depth_map_1[offset];
			}

			//没有深度则用最亮的深度值
			if(depth_map[offset]<= 0)
			{
				depth_map[offset] = depth_map_0[offset];
			}

		}
	}


__global__ void parallel_cuda_count_sum_pixel(const unsigned char* brightness,uint32_t img_height, uint32_t img_width, float* sum_pixels)
{
			const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
		const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
		const unsigned int offset = idy * img_width + idx;
	
		if (idx < img_width && idy < img_height)
		{ 
			*sum_pixels +=  brightness[offset];  
		}
}
	
bool parallel_cuda_copy_result_to_hdr(int serial_flag,int brigntness_serial)
{
	if(!load_calib_data_flag_)
	{
		return false;
	}
 

	CHECK(hipMemcpyAsync(d_hdr_depth_map_list_[serial_flag], d_depth_map_, 1 * image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToDevice)); 
	CHECK(hipMemcpyAsync(d_hdr_brightness_list_[serial_flag], d_patterns_list[brigntness_serial], 1 * image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToDevice));

	float val  = 0;
	CHECK(hipMemcpyAsync(d_hdr_bright_pixel_sum_list_[serial_flag], &val, sizeof(float), hipMemcpyHostToDevice)); 
 	parallel_cuda_count_sum_pixel << <blocksPerGrid, threadsPerBlock >> > (d_hdr_brightness_list_[serial_flag],image_height_,image_width_,d_hdr_bright_pixel_sum_list_[serial_flag]);
 
	LOG(INFO)<<"parallel_cuda_copy_result_to_hdr: "<<serial_flag;
	return true;
}



/***********************************************************************************************************************************************/

 bool cuda_merge_hdr_data(std::vector<float*> depth_map_list,std::vector<unsigned char*> brightness_list,float* depth_map, unsigned char* brightness)
{

	if(depth_map_list.size() != brightness_list.size())
	{
		return false;
	}

	LOG(INFO)<<"HDR";

	for(int i= 0;i< 3;i++)
	{ 
		CHECK(hipMemcpy(d_hdr_depth_map_list_[i], depth_map_list[i], image_height_*image_width_ * sizeof(float), hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_hdr_brightness_list_[i], brightness_list[i], image_height_*image_width_ * sizeof(unsigned char), hipMemcpyHostToDevice));
	}



	cuda_merge_hdr << <blocksPerGrid, threadsPerBlock >> > (d_hdr_depth_map_list_[0],d_hdr_depth_map_list_[1],d_hdr_depth_map_list_[2], d_hdr_brightness_list_[0], 
		d_hdr_brightness_list_[1], d_hdr_brightness_list_[2], image_height_, image_width_, d_hdr_depth_map_,d_hdr_brightness_);

	CHECK(hipMemcpy(depth_map, d_hdr_depth_map_, 1 * image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(brightness, d_hdr_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));


	LOG(INFO)<<"Finished!";


	return true;

}


bool cuda_get_frame_03_hdr(std::vector<unsigned char*> patterns,int group_flag,float* depth,unsigned char* brightness)
{
	if(31 != patterns.size())
	{
		return false;
	}

	if(!load_calib_data_flag_)
	{
		return false;
	}

	LOG(INFO)<<"hdr a: "<<group_flag;
	cuda_copy_patterns_hdr(patterns,group_flag); 

 
	LOG(INFO)<<"hdr b";
	cuda_compute_phase_03_hdr(group_flag); 
	 
	// LOG(INFO)<<"c";
	// cuda_unwrap_phase_03(); 
 

	// LOG(INFO)<<"d";
	// cuda_reconstruct();

	// LOG(INFO)<<"e";

	// cuda_get_brightness_data(brightness);

	// cuda_get_depth_data(depth);
 
 
	LOG(INFO)<<"f";
	return true;


}
	


bool cuda_get_frame_03(std::vector<unsigned char*> patterns,float* depth,unsigned char* brightness)
{
	if(31 != patterns.size())
	{
		return false;
	}

	if(!load_calib_data_flag_)
	{
		return false;
	}

	LOG(INFO)<<"a";
	cuda_copy_patterns(patterns); 

	LOG(INFO)<<"b";
	cuda_compute_phase_03();
//	CHECK(hipMemcpy(phase_map, d_wrap_map_list[0], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost));
	 
	LOG(INFO)<<"c";
	cuda_unwrap_phase_03(); 



	



	LOG(INFO)<<"d";
	cuda_reconstruct();

	LOG(INFO)<<"e";

	cuda_get_brightness_data(brightness);

	cuda_get_depth_data(depth);

	//cuda_get_frame_data(depth,brightness);


	LOG(INFO)<<"f";
	return true;


}
	


bool cuda_get_frame_base_24(std::vector<unsigned char*> patterns,float* depth,unsigned char* brightness)
{
	if(24 != patterns.size())
	{
		return false;
	}

	if(!load_calib_data_flag_)
	{
		return false;
	}

	LOG(INFO)<<"a";
	cuda_copy_patterns(patterns); 

	LOG(INFO)<<"b";
	cuda_compute_phase();
//	CHECK(hipMemcpy(phase_map, d_wrap_map_list[0], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost));
	 
	LOG(INFO)<<"c";
	cuda_unwrap_phase(); 



	



	LOG(INFO)<<"d";
	cuda_reconstruct();

	LOG(INFO)<<"e";
	
	cuda_get_frame_data(depth,brightness);


	LOG(INFO)<<"f";
	return true;


}
		

bool cuda_reconstruct_base_24(std::vector<unsigned char*> patterns, float* point_cloud)
{
	if(24 != patterns.size())
	{
		return false;
	}

	if(!load_calib_data_flag_)
	{
		return false;
	}

	LOG(INFO)<<"a";
	cuda_copy_patterns(patterns); 

	LOG(INFO)<<"b";
	cuda_compute_phase();
//	CHECK(hipMemcpy(phase_map, d_wrap_map_list[0], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost));
	 
	LOG(INFO)<<"c";
	cuda_unwrap_phase(); 
	 
	LOG(INFO)<<"d";
	cuda_reconstruct_pointcloud(point_cloud);

	LOG(INFO)<<"e";
	return true;
}


bool cuda_malloc_memory()
{ 
	
	for(int i= 0;i< patterns_count_;i++)
	{
		hipMalloc((void**)&d_patterns_list[i], image_height_*image_width_ * sizeof(unsigned char));
		
		// hipBindTexture(0,texture_patterns_list[i],d_patterns_list[i]);

		hipMalloc((void**)&d_patterns_list_hdr_0[i], image_height_*image_width_ * sizeof(unsigned char));
		hipMalloc((void**)&d_patterns_list_hdr_1[i], image_height_*image_width_ * sizeof(unsigned char));
		hipMalloc((void**)&d_patterns_list_hdr_2[i], image_height_*image_width_ * sizeof(unsigned char));
	}

	hipBindTexture(0,texture_patterns_0,d_patterns_list[0]);
	hipBindTexture(0,texture_patterns_1,d_patterns_list[1]);
	hipBindTexture(0,texture_patterns_2,d_patterns_list[2]);
	hipBindTexture(0,texture_patterns_3,d_patterns_list[3]);
	hipBindTexture(0,texture_patterns_4,d_patterns_list[4]);
	hipBindTexture(0,texture_patterns_5,d_patterns_list[5]);
	hipBindTexture(0,texture_patterns_6,d_patterns_list[6]);
	hipBindTexture(0,texture_patterns_7,d_patterns_list[7]);
	hipBindTexture(0,texture_patterns_8,d_patterns_list[8]);
	hipBindTexture(0,texture_patterns_9,d_patterns_list[9]);
	hipBindTexture(0,texture_patterns_10,d_patterns_list[10]);
	hipBindTexture(0,texture_patterns_11,d_patterns_list[11]);
	hipBindTexture(0,texture_patterns_12,d_patterns_list[12]);
	hipBindTexture(0,texture_patterns_13,d_patterns_list[13]);
	hipBindTexture(0,texture_patterns_14,d_patterns_list[14]);
	hipBindTexture(0,texture_patterns_15,d_patterns_list[15]);
	hipBindTexture(0,texture_patterns_16,d_patterns_list[16]);
	hipBindTexture(0,texture_patterns_17,d_patterns_list[17]);
	hipBindTexture(0,texture_patterns_18,d_patterns_list[18]);


 

	for (int i = 0; i< wrap_count_; i++)
	{
		hipMalloc((void**)&d_wrap_map_list[i], image_height_*image_width_ * sizeof(float));
		hipMalloc((void**)&d_confidence_list[i], image_height_*image_width_ * sizeof(float));

		hipMalloc((void**)&d_wrap_map_list_hdr_0[i], image_height_*image_width_ * sizeof(float));
		hipMalloc((void**)&d_confidence_list_hdr_0[i], image_height_*image_width_ * sizeof(float));
		hipMalloc((void**)&d_wrap_map_list_hdr_1[i], image_height_*image_width_ * sizeof(float));
		hipMalloc((void**)&d_confidence_list_hdr_1[i], image_height_*image_width_ * sizeof(float));
		hipMalloc((void**)&d_wrap_map_list_hdr_2[i], image_height_*image_width_ * sizeof(float));
		hipMalloc((void**)&d_confidence_list_hdr_2[i], image_height_*image_width_ * sizeof(float));
	}

	for (int i = 0; i< unwrap_count_; i++)
	{
		hipMalloc((void**)&d_unwrap_map_list[i], image_height_*image_width_ * sizeof(float));

		
		hipMalloc((void**)&d_unwrap_map_list_hdr_0[i], image_height_*image_width_ * sizeof(float)); 
		hipMalloc((void**)&d_unwrap_map_list_hdr_1[i], image_height_*image_width_ * sizeof(float)); 
		hipMalloc((void**)&d_unwrap_map_list_hdr_2[i], image_height_*image_width_ * sizeof(float));
	}

	for (int i = 0; i< D_HDR_MAX_NUM; i++)
	{
		hipMalloc((void**)&d_hdr_depth_map_list_[i], image_height_*image_width_ * sizeof(float));
		hipMalloc((void**)&d_hdr_brightness_list_[i], image_height_*image_width_ * sizeof(unsigned char)); 
		hipMalloc((void**)&d_hdr_bright_pixel_sum_list_[i], 1 * sizeof(float)); 
	}
	hipMalloc((void**)&d_hdr_depth_map_, image_height_*image_width_ * sizeof(float));
	hipMalloc((void**)&d_hdr_brightness_, image_height_*image_width_ * sizeof(unsigned char));


	hipMalloc((void**)&d_brightness_, image_height_*image_width_ * sizeof(unsigned char));
	hipMalloc((void**)&d_mask_, image_height_*image_width_ * sizeof(unsigned char));


	hipMalloc((void**)&d_camera_intrinsic_, 3*3 * sizeof(float));
	hipMalloc((void**)&d_project_intrinsic_, 3 * 3 * sizeof(float));

	hipMalloc((void**)&d_camera_distortion_, 1* 5 * sizeof(float));
	hipMalloc((void**)&d_projector_distortion_, 1 * 5 * sizeof(float));

	hipMalloc((void**)&d_rotation_matrix_, 3 * 3 * sizeof(float));
	hipMalloc((void**)&d_translation_matrix_, 1 * 3 * sizeof(float));


	hipMalloc((void**)&d_point_cloud_map_, 3*image_height_*image_width_ * sizeof(float));
	hipMalloc((void**)&d_depth_map_, image_height_*image_width_ * sizeof(float));
	hipMalloc((void**)&d_triangulation_error_map_, image_height_*image_width_ * sizeof(float));
 
	//分配重复patterns数据
	for(int i= 0;i< D_REPETITIONB_MAX_NUM*6;i++)
	{
		hipMalloc((void**)&d_repetition_patterns_list_[i], image_height_*image_width_ * sizeof(unsigned char)); 
	}

	for(int i= 0;i< 6;i++)
	{
		hipMalloc((void**)&d_repetition_merge_patterns_list_[i], image_height_*image_width_ * sizeof(unsigned short)); 
	}
 
 	for(int i= 0;i< D_REPETITION_02_MAX_NUM;i++)
	{
		hipMalloc((void**)&d_repetition_02_merge_patterns_list_[i], image_height_*image_width_ * sizeof(unsigned short)); 
	}
 
	
	
	reconstruct_cuda_malloc_memory();
	reconstruct_cuda_minimalloc_memory();

	hipDeviceSynchronize();

	return true;
}


bool cuda_free_memory()
{

	for (int i = 0; i< patterns_count_; i++)
	{ 
		// hipUnbindTexture(texture_patterns_list[i]);
		hipFree(d_patterns_list[i]);
		hipFree(d_patterns_list_hdr_0[i]);
		hipFree(d_patterns_list_hdr_1[i]);
		hipFree(d_patterns_list_hdr_2[i]);
	}

	hipUnbindTexture(texture_patterns_0);
	hipUnbindTexture(texture_patterns_1);
	hipUnbindTexture(texture_patterns_2);
	hipUnbindTexture(texture_patterns_3);
	hipUnbindTexture(texture_patterns_4);
	hipUnbindTexture(texture_patterns_5);
	hipUnbindTexture(texture_patterns_6);
	hipUnbindTexture(texture_patterns_7);
	hipUnbindTexture(texture_patterns_8);
	hipUnbindTexture(texture_patterns_9);
	hipUnbindTexture(texture_patterns_10);
	hipUnbindTexture(texture_patterns_11);
	hipUnbindTexture(texture_patterns_12);
	hipUnbindTexture(texture_patterns_13);
	hipUnbindTexture(texture_patterns_14);
	hipUnbindTexture(texture_patterns_15);
	hipUnbindTexture(texture_patterns_16);
	hipUnbindTexture(texture_patterns_17);
	hipUnbindTexture(texture_patterns_18);

	for (int i = 0; i< wrap_count_; i++)
	{ 
		
		hipFree(d_wrap_map_list[i]);
		hipFree(d_confidence_list[i]);

		hipFree(d_wrap_map_list_hdr_0[i]);
		hipFree(d_confidence_list_hdr_0[i]);

		hipFree(d_wrap_map_list_hdr_1[i]);
		hipFree(d_confidence_list_hdr_1[i]);

		hipFree(d_wrap_map_list_hdr_2[i]);
		hipFree(d_confidence_list_hdr_2[i]);

	}

	for (int i = 0; i< unwrap_count_; i++)
	{ 
		hipFree(d_unwrap_map_list[i]);
		
		hipFree(d_unwrap_map_list_hdr_0[i]); 
		hipFree(d_unwrap_map_list_hdr_1[i]);
		hipFree(d_unwrap_map_list_hdr_2[i]);
	}

	for (int i = 0; i< D_HDR_MAX_NUM; i++)
	{ 
		hipFree(d_hdr_depth_map_list_[i]);
		hipFree(d_hdr_brightness_list_[i]);
		hipFree(d_hdr_bright_pixel_sum_list_[i]);
	}
		hipFree(d_hdr_depth_map_);
		hipFree(d_hdr_brightness_);
	


	hipFree(d_brightness_);
	hipFree(d_mask_);

	hipFree(d_camera_intrinsic_);
	hipFree(d_project_intrinsic_);

	hipFree(d_camera_distortion_);
	hipFree(d_projector_distortion_);

	hipFree(d_rotation_matrix_);
	hipFree(d_translation_matrix_);


	hipFree(d_point_cloud_map_);
	hipFree(d_depth_map_);
	hipFree(d_triangulation_error_map_);

		//分配重复patterns数据
	for(int i= 0;i< D_REPETITIONB_MAX_NUM*6;i++)
	{
		hipFree(d_repetition_patterns_list_[i]); 
	}

	for(int i= 0;i< 6;i++)
	{
		hipFree(d_repetition_merge_patterns_list_[i]);  
	}

	for(int i= 0;i< D_REPETITION_02_MAX_NUM;i++)
	{
		hipFree(d_repetition_02_merge_patterns_list_[i]);  
	}

	
 
	reconstruct_cuda_free_memory();
	reconstruct_cuda_free_minimemory();

	return true;
}

bool cuda_reconstruct()
{

	cuda_rebuild << <blocksPerGrid, threadsPerBlock >> >(d_unwrap_map_list[0], d_unwrap_map_list[1],d_camera_intrinsic_,d_camera_distortion_,
		d_project_intrinsic_,d_projector_distortion_,d_rotation_matrix_,d_translation_matrix_, 
		d_point_cloud_map_,d_depth_map_, d_triangulation_error_map_, d_confidence_list[2]);


	hipDeviceSynchronize();
    LOG(INFO)<<"rebuild data!";
	
 

    // LOG(INFO)<<"unwrap_0";

	// cv::Mat unwrap_0(image_height_, image_width_, CV_32F, cv::Scalar(0));
	// hipMemcpy(unwrap_0.data, d_unwrap_map_list[0], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);

	// cv::Mat unwrap_1(image_height_, image_width_, CV_32F, cv::Scalar(0));
	// hipMemcpy(unwrap_1.data, d_unwrap_map_list[1], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);

	// cv::Mat deep_map(image_height_, image_width_, CV_32F, cv::Scalar(0));
	// hipMemcpy(deep_map.data, d_depth_map_, image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);

	// cv::Mat err_map(image_height_, image_width_, CV_32F, cv::Scalar(0));
	// hipMemcpy(err_map.data, d_triangulation_error_map_, image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);

	// cv::Mat points_map(image_height_, image_width_, CV_32FC3, cv::Scalar(0));
	// hipMemcpy(points_map.data, d_point_cloud_map_, 3*image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);

 
	
    // LOG(INFO)<<"copy data!";

	// cv::imwrite("unwrap_map_0.tiff",unwrap_0);
	// cv::imwrite("unwrap_map_1.tiff",unwrap_1);
	// cv::imwrite("deep_map.tiff",deep_map);
	// cv::imwrite("err_map.tiff",err_map);
	// cv::imwrite("points_map.tiff",points_map);


}


bool cuda_get_pointcloud_data(float* pointcloud)
{
	
	CHECK(hipMemcpy(pointcloud, d_point_cloud_map_, 3 * image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost));
}

bool cuda_get_depth_data(float* depth)
{


	CHECK(hipMemcpy(depth, d_depth_map_, 1 * image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost));

}

bool cuda_get_brightness_data(unsigned char* brightness)
{


	CHECK(hipMemcpy(brightness, d_patterns_list[30], 1 * image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));

}



bool cuda_get_frame_data(float* depth,unsigned char* bright)
{


	cuda_merge_brightness << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list[0], d_patterns_list[1], d_patterns_list[2],
			d_patterns_list[3], image_height_, image_width_, d_brightness_);


	hipDeviceSynchronize();
	CHECK(hipMemcpy(depth, d_depth_map_, 1 * image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(bright, d_brightness_, 1*image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost));

}

bool cuda_reconstruct_pointcloud(float* point_cloud)
{
	cuda_rebuild << <blocksPerGrid, threadsPerBlock >> >(d_unwrap_map_list[0], d_unwrap_map_list[1],d_camera_intrinsic_,d_camera_distortion_,
		d_project_intrinsic_,d_projector_distortion_,d_rotation_matrix_,d_translation_matrix_, 
		d_point_cloud_map_,d_depth_map_, d_triangulation_error_map_, d_confidence_list[2]);


	hipDeviceSynchronize();

	CHECK(hipMemcpy(point_cloud, d_point_cloud_map_, 3 * image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost));

	//cv::Mat test_point_cloud(image_height_, image_width_, CV_32FC3, cv::Scalar(0));
	//hipMemcpy(test_point_cloud.data, d_cuda_point_cloud_map_, 3*image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);

	//point_cloud = test_point_cloud.clone();

	//cv::Mat test_depth(image_height_, image_width_, CV_32F, cv::Scalar(0));
	//hipMemcpy(test_depth.data, d_cuda_depth_map_, image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);

	//cv::Mat test_error(image_height_, image_width_, CV_32F, cv::Scalar(0));
	//hipMemcpy(test_error.data, d_cuda_triangulation_error_map_, image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);

	return true;
}



bool cuda_unwrap_phase_03()
{
	
	 
	cuda_mul_phase_unwrap << <blocksPerGrid, threadsPerBlock >> >(d_wrap_map_list[0], d_wrap_map_list[1], d_wrap_map_list[2],image_height_, image_width_, d_unwrap_map_list[0]);

	// hipDeviceSynchronize();


	
	cuda_variable_phase_unwrap << <blocksPerGrid, threadsPerBlock >> >(d_unwrap_map_list[0], d_wrap_map_list[3], 4.0,
			image_height_, image_width_, 1.5,d_unwrap_map_list[0]);
	

	
	cuda_mul_phase_unwrap << <blocksPerGrid, threadsPerBlock >> >(d_wrap_map_list[4], d_wrap_map_list[5], d_wrap_map_list[6],image_height_, image_width_, d_unwrap_map_list[1]);


	cuda_normalize_phase << <blocksPerGrid, threadsPerBlock >> >(d_unwrap_map_list[0],128.0, d_unwrap_map_list[1],18.0,
			image_height_, image_width_, d_unwrap_map_list[0],d_unwrap_map_list[1]);
	


//	cv::Mat test_unwrap_x(image_height_, image_width_, CV_32F, cv::Scalar(0));
//	cv::Mat test_unwrap_y(image_height_, image_width_, CV_32F, cv::Scalar(0));
//	hipMemcpy(test_unwrap_x.data, d_unwrap_map_list[0], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);
//	hipMemcpy(test_unwrap_y.data, d_unwrap_map_list[1], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);
//	std::string path_x = "../../../debug_data/unwrap_img_x.tiff";
//	cv::imwrite(path_x,test_unwrap_x);
//	std::string path_y = "../../../debug_data/unwrap_img_y.tiff";
//	cv::imwrite(path_y,test_unwrap_y);
	

	//cv::Mat test_memory(image_height_, image_width_, CV_32F, cv::Scalar(0));
	//hipMemcpy(test_memory.data, d_cuda_unwrap_map_list[1], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);

	return true;
}


bool cuda_unwrap_phase()
{
	
	for(int i= 0;i< unwrap_count_;i++)
	{ 
		cuda_mul_phase_unwrap << <blocksPerGrid, threadsPerBlock >> >(d_wrap_map_list[3*i+0], d_wrap_map_list[3 * i + 1], d_wrap_map_list[3 * i + 2],
			image_height_, image_width_, d_unwrap_map_list[i]);
	}
	
	cuda_normalize_phase << <blocksPerGrid, threadsPerBlock >> >(d_unwrap_map_list[0],32.0, d_unwrap_map_list[1],32.0*720/1280,
			image_height_, image_width_, d_unwrap_map_list[0],d_unwrap_map_list[1]);
	

	


//	cv::Mat test_unwrap_x(image_height_, image_width_, CV_32F, cv::Scalar(0));
//	cv::Mat test_unwrap_y(image_height_, image_width_, CV_32F, cv::Scalar(0));
//	hipMemcpy(test_unwrap_x.data, d_unwrap_map_list[0], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);
//	hipMemcpy(test_unwrap_y.data, d_unwrap_map_list[1], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);
//	std::string path_x = "../../../debug_data/unwrap_img_x.tiff";
//	cv::imwrite(path_x,test_unwrap_x);
//	std::string path_y = "../../../debug_data/unwrap_img_y.tiff";
//	cv::imwrite(path_y,test_unwrap_y);
	

	//cv::Mat test_memory(image_height_, image_width_, CV_32F, cv::Scalar(0));
	//hipMemcpy(test_memory.data, d_cuda_unwrap_map_list[1], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);

	return true;
}



bool cuda_compute_phase_03_hdr(int group_flag)
{
	 
	
	switch(group_flag)
	{

		case 0:
		{
			for(int i= 0;i< 3;i++)
			{ 
				cuda_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list_hdr_0[4*i+0], d_patterns_list_hdr_0[4 * i + 1], d_patterns_list_hdr_0[4 * i + 2],
					d_patterns_list_hdr_0[4 * i + 3], image_height_, image_width_, d_wrap_map_list_hdr_0[i], d_confidence_list_hdr_0[i]);
			}
		 
			for(int i= 3;i< 4;i++)
			{
		
				cuda_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list_hdr_0[4*i+0], d_patterns_list_hdr_0[4 * i + 1], d_patterns_list_hdr_0[4 * i + 2],
					d_patterns_list_hdr_0[4 * i + 3],d_patterns_list_hdr_0[4 * i + 4],d_patterns_list_hdr_0[4 * i + 5] , image_height_, image_width_, d_wrap_map_list_hdr_0[i], d_confidence_list_hdr_0[i]);
			}
		
			int offset_i = 2;
		
			for(int i= 4;i< 7;i++)
			{ 
				cuda_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list_hdr_0[4*i + offset_i +0], d_patterns_list_hdr_0[4 * i + offset_i + 1], d_patterns_list_hdr_0[4 * i + offset_i + 2],
					d_patterns_list_hdr_0[4 * i + offset_i + 3], image_height_, image_width_, d_wrap_map_list_hdr_0[i], d_confidence_list_hdr_0[i]);
			}
			
		}
		break;

		case 1:
		{

			for(int i= 0;i< 3;i++)
			{ 
				cuda_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list_hdr_1[4*i+0], d_patterns_list_hdr_1[4 * i + 1], d_patterns_list_hdr_1[4 * i + 2],
					d_patterns_list_hdr_1[4 * i + 3], image_height_, image_width_, d_wrap_map_list_hdr_1[i], d_confidence_list_hdr_1[i]);
			}
		 
			for(int i= 3;i< 4;i++)
			{
		
				cuda_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list_hdr_1[4*i+0], d_patterns_list_hdr_1[4 * i + 1], d_patterns_list_hdr_1[4 * i + 2],
					d_patterns_list_hdr_1[4 * i + 3],d_patterns_list_hdr_1[4 * i + 4],d_patterns_list_hdr_1[4 * i + 5] , image_height_, image_width_, d_wrap_map_list_hdr_1[i], d_confidence_list_hdr_1[i]);
			}
		
			int offset_i = 2;
		
			for(int i= 4;i< 7;i++)
			{ 
				cuda_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list_hdr_1[4*i + offset_i +0], d_patterns_list_hdr_1[4 * i + offset_i + 1], d_patterns_list_hdr_1[4 * i + offset_i + 2],
					d_patterns_list_hdr_1[4 * i + offset_i + 3], image_height_, image_width_, d_wrap_map_list_hdr_1[i], d_confidence_list_hdr_1[i]);
			}

		}
		break;

		case 2:
		{ 		
			for(int i= 0;i< 3;i++)
			{ 
				cuda_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list_hdr_2[4*i+0], d_patterns_list_hdr_2[4 * i + 1], d_patterns_list_hdr_2[4 * i + 2],
					d_patterns_list_hdr_2[4 * i + 3], image_height_, image_width_, d_wrap_map_list_hdr_2[i], d_confidence_list_hdr_2[i]);
			}
		 
			for(int i= 3;i< 4;i++)
			{
		
				cuda_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list_hdr_2[4*i+0], d_patterns_list_hdr_2[4 * i + 1], d_patterns_list_hdr_2[4 * i + 2],
					d_patterns_list_hdr_2[4 * i + 3],d_patterns_list_hdr_2[4 * i + 4],d_patterns_list_hdr_2[4 * i + 5] , image_height_, image_width_, d_wrap_map_list_hdr_2[i], d_confidence_list_hdr_2[i]);
			}
		
			int offset_i = 2;
		
			for(int i= 4;i< 7;i++)
			{ 
				cuda_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list_hdr_2[4*i + offset_i +0], d_patterns_list_hdr_2[4 * i + offset_i + 1], d_patterns_list_hdr_2[4 * i + offset_i + 2],
					d_patterns_list_hdr_2[4 * i + offset_i + 3], image_height_, image_width_, d_wrap_map_list_hdr_2[i], d_confidence_list_hdr_2[i]);
			}

		}
		break;

		default:
		break;


	}










	// hipDeviceSynchronize();

	//cv::Mat test_patterns(image_height_, image_width_, CV_8U, cv::Scalar(0));
	//hipMemcpy(test_patterns.data,d_patterns_list[0], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);
//	std::string path = "../../debug_data/unwrap_img_0.tiff";
//	cv::imwrite(path,test_patterns);
		


//	cv::Mat test_memory(image_height_, image_width_, CV_32F, cv::Scalar(0));
//	hipMemcpy(test_memory.data, d_wrap_map_list[0], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);
//	std::string path = "../../debug_data/wrap_img_0.tiff";
//	cv::imwrite(path,test_memory);
	
	return true;
}



bool cuda_compute_phase_03()
{
	 
	for(int i= 0;i< 3;i++)
	{

		cuda_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list[4*i+0], d_patterns_list[4 * i + 1], d_patterns_list[4 * i + 2],
			d_patterns_list[4 * i + 3], image_height_, image_width_, d_wrap_map_list[i], d_confidence_list[i]);
	}


	for(int i= 3;i< 4;i++)
	{

		cuda_six_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list[4*i+0], d_patterns_list[4 * i + 1], d_patterns_list[4 * i + 2],
			d_patterns_list[4 * i + 3],d_patterns_list[4 * i + 4],d_patterns_list[4 * i + 5] , image_height_, image_width_, d_wrap_map_list[i], d_confidence_list[i]);
	}

	int offset_i = 2;

	for(int i= 4;i< 7;i++)
	{

		cuda_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list[4*i + offset_i +0], d_patterns_list[4 * i + offset_i + 1], d_patterns_list[4 * i + offset_i + 2],
			d_patterns_list[4 * i + offset_i + 3], image_height_, image_width_, d_wrap_map_list[i], d_confidence_list[i]);
	}




	// hipDeviceSynchronize();

	//cv::Mat test_patterns(image_height_, image_width_, CV_8U, cv::Scalar(0));
	//hipMemcpy(test_patterns.data,d_patterns_list[0], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);
//	std::string path = "../../debug_data/unwrap_img_0.tiff";
//	cv::imwrite(path,test_patterns);
		


//	cv::Mat test_memory(image_height_, image_width_, CV_32F, cv::Scalar(0));
//	hipMemcpy(test_memory.data, d_wrap_map_list[0], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);
//	std::string path = "../../debug_data/wrap_img_0.tiff";
//	cv::imwrite(path,test_memory);
	
	return true;
}



bool cuda_compute_phase()
{
	 
	for(int i= 0;i< wrap_count_;i++)
	{

		cuda_four_step_phase_shift << <blocksPerGrid, threadsPerBlock >> > (d_patterns_list[4*i+0], d_patterns_list[4 * i + 1], d_patterns_list[4 * i + 2],
			d_patterns_list[4 * i + 3], image_height_, image_width_, d_wrap_map_list[i], d_confidence_list[i]);
	}


	hipDeviceSynchronize();

	//cv::Mat test_patterns(image_height_, image_width_, CV_8U, cv::Scalar(0));
	//hipMemcpy(test_patterns.data,d_patterns_list[0], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);
//	std::string path = "../../debug_data/unwrap_img_0.tiff";
//	cv::imwrite(path,test_patterns);
		


//	cv::Mat test_memory(image_height_, image_width_, CV_32F, cv::Scalar(0));
//	hipMemcpy(test_memory.data, d_wrap_map_list[0], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost);
//	std::string path = "../../debug_data/wrap_img_0.tiff";
//	cv::imwrite(path,test_memory);
	
	return true;
}

bool cuda_copy_calib_data(float* camera_intrinsic, float* project_intrinsic, float* camera_distortion,
	float* projector_distortion, float* rotation_matrix, float* translation_matrix)
{
	//if(!camera_intrinsic.data || !project_intrinsic.data || !camera_distortion.data
	//	|| !projector_distortion.data || !rotation_matrix.data || !translation_matrix.data)
	//{
	//	return false;
	//}

	CHECK(hipMemcpy(d_camera_intrinsic_, camera_intrinsic, 3 * 3 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_project_intrinsic_, project_intrinsic, 3 * 3 * sizeof(float), hipMemcpyHostToDevice));

	CHECK(hipMemcpy(d_camera_distortion_, camera_distortion, 1 * 5 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_projector_distortion_, projector_distortion, 1 * 5 * sizeof(float), hipMemcpyHostToDevice));

	CHECK(hipMemcpy(d_rotation_matrix_, rotation_matrix, 3 * 3 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_translation_matrix_, translation_matrix, 1* 3 * sizeof(float), hipMemcpyHostToDevice));

	load_calib_data_flag_ = true;

	//cv::Mat test_memory(3, 3, CV_32F, cv::Scalar(0));
	//hipMemcpy(test_memory.data, camera_intrinsic_, 3*3 * sizeof(float), hipMemcpyDeviceToHost);

	return true;
}

bool cuda_copy_patterns_hdr(std::vector<unsigned char*> patterns,int flag)
{
	
	if(patterns.empty())
	{
		return false;
	}


	switch(flag)
	{

		case 0:
		{
			for(int i= 0;i< patterns.size();i++)
			{ 
				CHECK(hipMemcpy(d_patterns_list_hdr_0[i], patterns[i], image_height_*image_width_ * sizeof(unsigned char), hipMemcpyHostToDevice));
			}
		}
		break;

		case 1:
		{
			for(int i= 0;i< patterns.size();i++)
			{ 
				CHECK(hipMemcpy(d_patterns_list_hdr_1[i], patterns[i], image_height_*image_width_ * sizeof(unsigned char), hipMemcpyHostToDevice));
			}
		}
		break;

		case 2:
		{ 			for(int i= 0;i< patterns.size();i++)
			{ 
				CHECK(hipMemcpy(d_patterns_list_hdr_2[i], patterns[i], image_height_*image_width_ * sizeof(unsigned char), hipMemcpyHostToDevice));
			}
		}
		break;

		default:
		break;


	}



	   

	return true;
}

bool cuda_copy_patterns(std::vector<unsigned char*> patterns)
{
	
	if(patterns.empty())
	{
		return false;
	}

	for(int i= 0;i< patterns.size();i++)
	{ 
		CHECK(hipMemcpy(d_patterns_list[i], patterns[i], image_height_*image_width_ * sizeof(unsigned char), hipMemcpyHostToDevice));
	}
	  
	// hipDeviceSynchronize();

	return true;
}

__global__ void cuda_merge_hdr(const float*  depth_map_0,const float*  depth_map_1,const float*  depth_map_2,const unsigned char* brightness_0,const unsigned char* brightness_1,
	const unsigned char* brightness_2,uint32_t img_height, uint32_t img_width, float* const depth_map,unsigned char * const brightness)
{

	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * img_width + idx;

	if (idx < img_width && idy < img_height)
	{



		float pixel= 0;
		pixel +=  brightness_0[offset];
		pixel +=  brightness_1[offset];
		pixel +=  brightness_2[offset];

		pixel/= 3.0;


		brightness[offset] = pixel;

		if(brightness_0[offset] < 255)
		{
			// brightness[offset] = brightness_0[offset];
			depth_map[offset] = depth_map_0[offset];
		}

		else if(brightness_1[offset] < 255)
		{
			// brightness[offset] = brightness_1[offset];
			depth_map[offset] = depth_map_1[offset];
		}
		else
		{	
			// brightness[offset] = brightness_2[offset];
			depth_map[offset] = depth_map_2[offset];
		}


	}



	
}




__global__ void cuda_merge_brightness(unsigned char* const d_in_0,unsigned char* const d_in_1,unsigned char* d_in_2,unsigned char* d_in_3,
	uint32_t img_height, uint32_t img_width,unsigned char * const d_out)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * img_width + idx;

	if (idx < img_width && idy < img_height)
	{

		float a = d_in_3[offset] - d_in_1[offset];
		float b = d_in_0[offset] - d_in_2[offset];

		float ave = (d_in_0[offset] + d_in_1[offset] + d_in_2[offset] +d_in_3[offset])/4.0;
  
		float val = ave + std::sqrt(a*a + b*b);
 

		if(val> 255)
		{
		   val = 255.0;
		}
		unsigned char c = val;	
		d_out[offset] = c;
	}

	

}


__global__ void cuda_six_step_phase_shift_texture(float * const d_out, float * const confidence)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy *  d_image_width_ + idx;
	float s_0 =  0;
	float s_1 =  0.866025;
	float s_2 =  0.866025;
	float s_3 =  0;
	float s_4 =  -0.866025;
	float s_5 =  -0.866025;
	float c_0 =  1;
	float c_1 =  0.5;
	float c_2 =  -0.5;
	float c_3 =  -1;
	float c_4 =  -0.5;
	float c_5 =  0.5;

 

	unsigned char pixel_0 = tex1Dfetch(texture_patterns_12, offset);
	unsigned char pixel_1 = tex1Dfetch(texture_patterns_13, offset);
	unsigned char pixel_2 = tex1Dfetch(texture_patterns_14, offset);
	unsigned char pixel_3 = tex1Dfetch(texture_patterns_15, offset);
	unsigned char pixel_4 = tex1Dfetch(texture_patterns_16, offset);
	unsigned char pixel_5 = tex1Dfetch(texture_patterns_17, offset);

	if (idx < d_image_width_ && idy < d_image_height_)
	{
  
		float a = c_0 *pixel_3 + c_1 *pixel_4 + c_2 *pixel_5 + c_3* pixel_0 +c_4*pixel_1 + c_5*pixel_2;
		float b = s_0 *pixel_3 + s_1 *pixel_4 + s_2 *pixel_5 + s_3* pixel_0 +s_4*pixel_1 + s_5*pixel_2;


		int over_num = 0;
		if(pixel_0 >= 255)
		{
			over_num++;
		}
		if (pixel_1 >= 255)
		{
			over_num++;
		}
		if (pixel_2 >= 255)
		{
			over_num++;
		}
		if (pixel_3 >= 255)
		{
			over_num++;
		}
		if (pixel_4 >= 255)
		{
			over_num++;
		}
		if (pixel_5 >= 255)
		{
			over_num++;
		}

		if(over_num> 3)
		{
			confidence[offset] = 0;
			d_out[offset] = -1;
		}
		else
		{
			confidence[offset] = std::sqrt(a*a + b*b);
			d_out[offset] = DF_PI + std::atan2(a, b);
		}
  
		// confidence[offset] = std::sqrt(a*a + b*b);
		// d_out[offset] = DF_PI + std::atan2(a, b);
	}
}

__global__ void cuda_six_step_phase_shift(unsigned char * const d_in_0, unsigned char * const d_in_1, unsigned char * const d_in_2, unsigned char * const d_in_3,unsigned char* const d_in_4,unsigned char* const d_in_5,
	uint32_t img_height, uint32_t img_width,float * const d_out, float * const confidence)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * img_width + idx;
	float s_0 =  0;
	float s_1 =  0.866025;
	float s_2 =  0.866025;
	float s_3 =  0;
	float s_4 =  -0.866025;
	float s_5 =  -0.866025;
	float c_0 =  1;
	float c_1 =  0.5;
	float c_2 =  -0.5;
	float c_3 =  -1;
	float c_4 =  -0.5;
	float c_5 =  0.5;
	
	if (idx < img_width && idy < img_height)
	{

		float a = c_0 *d_in_3[offset] + c_1 *d_in_4[offset] + c_2 *d_in_5[offset] + c_3* d_in_0[offset] +c_4*d_in_1[offset] + c_5*d_in_2[offset];
		float b = s_0 *d_in_3[offset] + s_1 *d_in_4[offset] + s_2 *d_in_5[offset] + s_3* d_in_0[offset] +s_4*d_in_1[offset] + s_5*d_in_2[offset];


		int over_num = 0;
		if(d_in_0[offset]>= 255)
		{
			over_num++;
		}
		if (d_in_1[offset] >= 255)
		{
			over_num++;
		}
		if (d_in_2[offset] >= 255)
		{
			over_num++;
		}
		if (d_in_3[offset] >= 255)
		{
			over_num++;
		}
		if (d_in_4[offset] >= 255)
		{
			over_num++;
		}
		if (d_in_5[offset] >= 255)
		{
			over_num++;
		}

		if(over_num> 3)
		{
			confidence[offset] = 0;
			d_out[offset] = -1;
		}
		else
		{
			confidence[offset] = std::sqrt(a*a + b*b);
			d_out[offset] = DF_PI + std::atan2(a, b);
		}
  
		// confidence[offset] = std::sqrt(a*a + b*b);
		// d_out[offset] = DF_PI + std::atan2(a, b);
	}
}



__global__ void cuda_four_step_phase_shift_texture(int serial_flag,float * const d_out, float * const confidence)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * d_image_width_ + idx;
	unsigned char pixel_0 = 0;
	unsigned char pixel_1 = 0;
	unsigned char pixel_2 = 0;
	unsigned char pixel_3 = 0;

	if (idx < d_image_width_ && idy < d_image_height_)
	{


		switch (serial_flag)
		{
		case 0:
		{
			pixel_0 = tex1Dfetch(texture_patterns_0, offset);
			pixel_1 = tex1Dfetch(texture_patterns_1, offset);
			pixel_2 = tex1Dfetch(texture_patterns_2, offset);
			pixel_3 = tex1Dfetch(texture_patterns_3, offset);
		}
		break;
		case 1:
		{
			pixel_0 = tex1Dfetch(texture_patterns_4, offset);
			pixel_1 = tex1Dfetch(texture_patterns_5, offset);
			pixel_2 = tex1Dfetch(texture_patterns_6, offset);
			pixel_3 = tex1Dfetch(texture_patterns_7, offset);
		}
		break;
		case 2:
		{
			pixel_0 = tex1Dfetch(texture_patterns_8, offset);
			pixel_1 = tex1Dfetch(texture_patterns_9, offset);
			pixel_2 = tex1Dfetch(texture_patterns_10, offset);
			pixel_3 = tex1Dfetch(texture_patterns_11, offset);
		}
		break;

		default:
			break;
		}



		float a = pixel_3 - pixel_1;
		float b = pixel_0 - pixel_2;

		int over_num = 0;
		if(pixel_0 >= 255)
		{
			over_num++;
		}
		if (pixel_1 >= 255)
		{
			over_num++;
		}
		if (pixel_2 >= 255)
		{
			over_num++;
		}
		if (pixel_3 >= 255)
		{
			over_num++;
		}

		if(over_num> 1)
		{
			confidence[offset] = 0;
			d_out[offset] = -1;
		}
		else
		{
			confidence[offset] = std::sqrt(a*a + b*b);
			d_out[offset] = DF_PI + std::atan2(a, b);
		}





	}
}


__global__ void cuda_merge_brigntness_map(unsigned short * const merge_brightness,int repetition_count, unsigned char* brightness)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * d_image_width_ + idx;
 

	if (idx < d_image_width_ && idy < d_image_height_)
	{ 
		brightness[offset] = 0.5 + (merge_brightness[offset]/repetition_count); 
  	 
	}
}

__global__ void cuda_merge_four_step_phase_shift(unsigned short * const d_in_0, unsigned short * const d_in_1, unsigned short * const d_in_2, 
	unsigned short * const d_in_3,int repetition_count,float * const d_out, float * const confidence)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * d_image_width_ + idx;

	int max_pixel = 255*repetition_count;

	if (idx < d_image_width_ && idy < d_image_height_)
	{

		float a = d_in_3[offset] - d_in_1[offset];
		float b = d_in_0[offset] - d_in_2[offset];

		int over_num = 0;
		if(d_in_0[offset]>= max_pixel)
		{
			over_num++;
		}
		if (d_in_1[offset] >= max_pixel)
		{
			over_num++;
		}
		if (d_in_2[offset] >= max_pixel)
		{
			over_num++;
		}
		if (d_in_3[offset] >= max_pixel)
		{
			over_num++;
		}

		if(over_num> 1)
		{
			confidence[offset] = 0;
			d_out[offset] = -1;
		}
		else
		{
			confidence[offset] = std::sqrt(a*a + b*b);
			d_out[offset] = DF_PI + std::atan2(a, b);
		}
  
	}
}

__global__ void cuda_four_step_phase_shift(unsigned char * const d_in_0, unsigned char * const d_in_1, unsigned char * const d_in_2, unsigned char * const d_in_3,
	uint32_t img_height, uint32_t img_width,float * const d_out, float * const confidence)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * img_width + idx;

	if (idx < img_width && idy < img_height)
	{

		float a = d_in_3[offset] - d_in_1[offset];
		float b = d_in_0[offset] - d_in_2[offset];

		int over_num = 0;
		if(d_in_0[offset]>= 255)
		{
			over_num++;
		}
		if (d_in_1[offset] >= 255)
		{
			over_num++;
		}
		if (d_in_2[offset] >= 255)
		{
			over_num++;
		}
		if (d_in_3[offset] >= 255)
		{
			over_num++;
		}

		if(over_num> 1)
		{
			confidence[offset] = 0;
			d_out[offset] = -1;
		}
		else
		{
			confidence[offset] = std::sqrt(a*a + b*b);
			d_out[offset] = DF_PI + std::atan2(a, b);
		}





	}
}


__global__ void cuda_normalize_phase(float * const d_in_unwrap_x, float rate_x,float * const d_in_unwrap_y,float rate_y,uint32_t img_height, uint32_t img_width, float * const d_out_normal_x,float * d_out_normal_y)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

	const unsigned int offset = idy*img_width + idx;

	if (idx < img_width && idy < img_height)
	{

		/*****************************************************************************/

		d_out_normal_x[offset] = d_in_unwrap_x[offset] /rate_x;
		d_out_normal_y[offset] = d_in_unwrap_y[offset] /rate_y;
		

		/******************************************************************/
	}
}

__global__ void cuda_variable_phase_unwrap(float * const d_in_wrap_abs, float * const d_in_wrap_high,float const rate,uint32_t img_height, uint32_t img_width,float threshold, float * const d_out)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

	int offset = idy * img_width + idx;

	if (idx < img_width && idy < img_height)
	{

		/*****************************************************************************/

		float temp = 0.5 + (rate * d_in_wrap_abs[idy * img_width + idx] - d_in_wrap_high[idy * img_width + idx]) / (2*DF_PI);
		int k = temp;
		// d_out[idy * img_width + idx] = DF_PI*k + d_in_wrap_high[idy * img_width + idx];


		float unwrap_value =  2*DF_PI*k + d_in_wrap_high[idy * img_width + idx]; 
		float err = unwrap_value - (rate * d_in_wrap_abs[idy * img_width + idx]);
		if(abs(err)> threshold)
		{
			d_out[idy * img_width + idx] = -10.0; 
		}
		else
		{ 
			d_out[idy * img_width + idx] = unwrap_value;
		}

		/******************************************************************/
	}
}




 
__global__ void cuda_mul_phase_unwrap(float * const d_in_wrap_0, float * const d_in_wrap_1, float * const d_in_wrap_2,
	uint32_t img_height, uint32_t img_width, float * const d_out)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

	int offset = idy* img_width +idx;

	if (idx < img_width && idy < img_height)
	{

		/*****************************************************************************/

		double temp = 0.5 + (8 * d_in_wrap_0[idy * img_width + idx] - d_in_wrap_1[idy * img_width + idx]) / (DF_PI);
		int k = temp;
		
		
		d_out[idy * img_width + idx] = DF_PI*k + d_in_wrap_1[idy * img_width + idx];

		float err = d_out[offset] - 8 *d_in_wrap_0[offset];
		if(abs(err)> 1.0)
		{
			d_out[offset] = -10;
		}
		
		float old_val = d_out[offset];
		/******************************************************************/
		temp = 0.5 + (4 * d_out[idy * img_width + idx] - d_in_wrap_2[idy * img_width + idx]) / (DF_PI);
		k = temp;

		d_out[idy * img_width + idx] = DF_PI*k + d_in_wrap_2[idy * img_width + idx];

	        err = d_out[offset] - 4 * old_val;
		if(abs(err)> 1.0)
		{
			d_out[offset] = -10;
		}
		
	}
}


__device__ void triangulation(float x_norm_L, float y_norm_L, float x_norm_R, float y_norm_R, float* R, float* T,
	float& X_L, float& Y_L, float& Z_L, float& X_R, float& Y_R, float& Z_R,
	float& error)
{
	float u_x_L = R[0] * x_norm_L + R[1] * y_norm_L + R[2];
	float u_y_L = R[3] * x_norm_L + R[4] * y_norm_L + R[5];
	float u_w_L = R[6] * x_norm_L + R[7] * y_norm_L + R[8];

	float n_x2_L = x_norm_L * x_norm_L + y_norm_L * y_norm_L + 1;
	float n_x2_R = x_norm_R * x_norm_R + y_norm_R * y_norm_R + 1;

	float D = u_x_L * x_norm_R + u_y_L * y_norm_R + u_w_L;
	float DD = n_x2_L * n_x2_R - D * D;

	float dot_uT = u_x_L * T[0] + u_y_L * T[1] + u_w_L * T[2];
	float dot_xttT = x_norm_R * T[0] + y_norm_R * T[1] + T[2];
	float dot_xttu = u_x_L * x_norm_R + u_y_L * y_norm_R + u_w_L;

	float NN1 = dot_xttu*dot_xttT - n_x2_R*dot_uT;
	float NN2 = n_x2_L*dot_xttT - dot_uT*dot_xttu;

	float Zt = NN1 / DD;
	float Ztt = NN2 / DD;

	float X1 = x_norm_L * Zt;
	float Y1 = y_norm_L * Zt;
	float Z1 = Zt;

	float X2_R = x_norm_R * Ztt - T[0];
	float Y2_R = y_norm_R * Ztt - T[1];
	float Z2_R = Ztt - T[2];

	float X2 = R[0] * X2_R + R[3] * Y2_R + R[6] * Z2_R;
	float Y2 = R[1] * X2_R + R[4] * Y2_R + R[7] * Z2_R;
	float Z2 = R[2] * X2_R + R[5] * Y2_R + R[8] * Z2_R;

	X_L = (X1 + X2) / 2.0;
	Y_L = (Y1 + Y2) / 2.0;
	Z_L = (Z1 + Z2) / 2.0;

	//XR = R * XL + T;
	X_R = R[0] * X_L + R[1] * Y_L + R[2] * Z_L + T[0];
	Y_R = R[3] * X_L + R[4] * Y_L + R[5] * Z_L + T[1];
	Z_R = R[6] * X_L + R[7] * Y_L + R[8] * Z_L + T[2];

	error = sqrt((X1 - X2) * (X1 - X2) + (Y1 - Y2) * (Y1 - Y2) + (Z1 - Z2) * (Z1 - Z2));
	 
}

__device__ void  normalizePoint(float x, float y, float fc_x, float fc_y,
	float cc_x, float cc_y, float k1, float k2,  float p1, float p2, float k3,
	float& x_norm, float& y_norm)
{
	float x_distort = (x - cc_x) / fc_x;
	float y_distort = (y - cc_y) / fc_y;

	float x_iter = x_distort;
	float y_iter = y_distort;

	for (int i = 0; i < 20; i++)
	{
		float r_2 = x_iter * x_iter + y_iter * y_iter;
		float r_4 = r_2 * r_2;
		float r_6 = r_4 * r_2;
		float k_radial = 1 + k1 * r_2 + k2 * r_4 + k3 * r_6;
		float delta_x = 2 * p1 * x_iter * y_iter + p2 * (r_2 + 2 * x_iter * x_iter);
		float delta_y = p1 * (r_2 + 2 * y_iter * y_iter) + 2 * p2 * x_iter * y_iter;
		x_iter = (x_distort - delta_x) / k_radial;
		y_iter = (y_distort - delta_y) / k_radial;
	}
	//x_norm = x_iter*fc_x+ cc_x;
	//y_norm = y_iter*fc_y + cc_y;

	x_norm = x_iter;
	y_norm = y_iter;
}

__global__ void cuda_rebuild(float * const d_in_unwrap_x, float * const d_in_unwrap_y, float * const camera_intrinsic, float * const camera_distortion,
	 float * const projector_intrinsic, float * const projector_distortion, float * const rotation_matrix, float * const translation_matrix,
	float * const d_out_point_cloud_map, float * const d_out_depth_map, float * const d_out_error_map, float * const confidence_map)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;


	const unsigned int serial_id = idy * d_image_width_ + idx;

	if (idx < d_image_width_ && idy < d_image_height_)
	{
		/****************************************************************************/
		//phase to position
		float dlp_x = d_in_unwrap_x[idy * d_image_width_ + idx] * d_dlp_width_ / (128.0*2*DF_PI);
		float dlp_y = d_in_unwrap_y[idy * d_image_width_ + idx] * d_dlp_height_ / (18.0*2*DF_PI);

		//if(100 == idx && 100 == idy)
		//{
		//	printf("%f\n", camera_intrinsic[0]); 
		//}

		/*****************************************************************************/
		////undistort
		////fc_x = i[0];fc_y = i[4]; cc_x = i[2]; cc_y = i[5];
		////k1= d[0]; k2= d[1]; p1 = d[2]; p2 = d[3]; k3 = d[4] 
		 
		float x_norm_L = 0;
		float y_norm_L = 0;

		normalizePoint(idx, idy, camera_intrinsic[0], camera_intrinsic[4], camera_intrinsic[2], camera_intrinsic[5],
			camera_distortion[0], camera_distortion[1], camera_distortion[2], camera_distortion[3], camera_distortion[4],
			x_norm_L, y_norm_L);


		float x_norm_R = 0;
		float y_norm_R = 0;

		normalizePoint(dlp_x, dlp_y, projector_intrinsic[0], projector_intrinsic[4], projector_intrinsic[2], projector_intrinsic[5],
			projector_distortion[0], projector_distortion[1], projector_distortion[2], projector_distortion[3], projector_distortion[4],
			x_norm_R, y_norm_R);

 

		/**********************************************************************************************************/
		//reconstruct
		float X_L, Y_L, Z_L, X_R, Y_R, Z_R, error;

		triangulation(x_norm_L, y_norm_L, x_norm_R, y_norm_R, rotation_matrix, translation_matrix,
			X_L, Y_L, Z_L, X_R, Y_R, Z_R, error);
		if(confidence_map[serial_id] > 10 && error< 3.0 && dlp_x > 0 && dlp_y > 0)	
		//if(confidence_map[serial_id] > 10 && error< 0.5 && dlp_x> 0.0 && dlp_y > 0.0)
		{
		    d_out_point_cloud_map[3 * serial_id + 0] = X_L;
		    d_out_point_cloud_map[3 * serial_id + 1] = Y_L;
		    d_out_point_cloud_map[3 * serial_id + 2] = Z_L;
		    d_out_depth_map[serial_id] = Z_L;
		}
		else
		{
		    d_out_point_cloud_map[3 * serial_id + 0] = 0;
		    d_out_point_cloud_map[3 * serial_id + 1] = 0;
		    d_out_point_cloud_map[3 * serial_id + 2] = 0;
		    d_out_depth_map[serial_id] = 0;
		}

		d_out_error_map[serial_id] = error;


		/******************************************************************/
 

	}
}




/***************************************************************************************************/
__device__ float computePointsDistance(float* p0,float* p1)
{
	return std::sqrt(p0[0]*p1[0] + p0[1] *p1[1] + p0[2] * p1[2]);
}
 
//滤波
__global__ void cuda_removal_points_base_mask(uint32_t img_height, uint32_t img_width,float* const point_cloud_map,float* const deep_map,uchar* remove_mask)
{
  	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y; 
  
	const unsigned int serial_id = idy * img_width + idx;

	if (idx < img_width && idy < img_height)
	{
		if(0 == remove_mask[serial_id])
		{
			deep_map[serial_id] = 0;
			point_cloud_map[3 * serial_id + 0] = 0;
			point_cloud_map[3 * serial_id + 1] = 0;
			point_cloud_map[3 * serial_id + 2] = 0;
		}

	}

}

//滤波
__global__ void cuda_filter_radius_outlier_removal(uint32_t img_height, uint32_t img_width,float* const point_cloud_map,uchar* remove_mask,float dot_spacing, float radius,int threshold)
{
 	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y; 
  
	const unsigned int serial_id = idy * img_width + idx;

	if (idx < img_width && idy < img_height)
	{
		/****************************************************************************/
		//定位区域
		if (point_cloud_map[3 * serial_id + 2] > 0)
		{
			remove_mask[serial_id] = 255;

			int w = 1 + radius / dot_spacing;

			int s_r = idy - w;
			int s_c = idx - w;

			int e_r = idy + w;
			int e_c = idx + w;

			if (s_r < 0)
			{
				s_r = 0;
			}
			if (s_c < 0)
			{
				s_c = 0;
			}

			if (e_r >= img_height)
			{
				e_r = img_height - 1;
			}

			if (e_c >= img_width)
			{
				e_c = img_width - 1;
			}

			int num = 0;

			for (int r = s_r; r <= e_r; r++)
			{
				for (int c = s_c; c <= e_c; c++)
				{
					int pos = r * img_width + c;
					if (point_cloud_map[3 * pos + 2] > 0)
					{  
						float dx= point_cloud_map[3 * serial_id + 0] - point_cloud_map[3 * pos + 0];
						float dy= point_cloud_map[3 * serial_id + 1] - point_cloud_map[3 * pos + 1];
						float dz= point_cloud_map[3 * serial_id + 2] - point_cloud_map[3 * pos + 2];

						float dist = std::sqrt(dx * dx + dy * dx + dz * dz); 
 
						if (radius > dist)
						{
							num++;
						}
					}
				}
			} 

			if (num < threshold)
			{ 
				remove_mask[serial_id] = 0;
			} 
		}
		else
		{ 
			remove_mask[serial_id] = 0;
		}

		/******************************************************************/
	}
}


/*****************************************************************************************************************************************************/


bool generate_pointcloud_base_table()
{

	reconstruct_pointcloud_base_table << <blocksPerGrid, threadsPerBlock >> > (d_xL_rotate_x_ , d_xL_rotate_y_, 
                                                d_single_pattern_mapping_, d_R_1_,d_confidence_list[3],d_unwrap_map_list[0],
												image_height_,image_width_,d_baseline_,d_point_cloud_map_,d_depth_map_);


	// LOG(INFO)<<"remove start:";
	// //相机像素为5.4um、焦距12mm。dot_spacing = 5.4*distance/12000 mm，典型值0.54mm（1200） 
	cuda_filter_radius_outlier_removal << <blocksPerGrid, threadsPerBlock >> > (image_height_,image_width_,d_point_cloud_map_,d_mask_,0.5,2.5,6); 
	cuda_removal_points_base_mask << <blocksPerGrid, threadsPerBlock >> > (image_height_,image_width_,d_point_cloud_map_,d_depth_map_,d_mask_); 

    // hipDeviceSynchronize();
	// cv::Mat point_cloud_map(1200, 1920, CV_32FC3, cv::Scalar(0.0));  
	// CHECK(hipMemcpy((float*)point_cloud_map.data, d_point_cloud_map_, 3 * image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost));

	// std::vector<cv::Mat> channel;
	// cv::split(point_cloud_map, channel); 
	// cv::imwrite("pointcloud_map_x.tiff",channel[0]);
	// cv::imwrite("pointcloud_map_y.tiff",channel[1]);
	// cv::imwrite("pointcloud_map_z.tiff",channel[2]);
	// LOG(INFO)<<"remove finished!";
}
 
 bool generate_pointcloud_base_minitable()
{

	reconstruct_pointcloud_base_minitable << <blocksPerGrid, threadsPerBlock >> > (d_xL_rotate_x_, d_xL_rotate_y_,
		d_single_pattern_minimapping_, d_R_1_, d_confidence_list[3], d_unwrap_map_list[0],
		image_height_, image_width_, d_baseline_, d_point_cloud_map_, d_depth_map_);


}

__device__ float bilinear_interpolation(float x, float y, int map_width, float *mapping)
{

	int x1 = floor(x);
	int y1 = floor(y);
	int x2 = x1 + 1;
	int y2 = y1 + 1;

	//row-y,col-x

	if (x1 == 1919) {
		float out = mapping[y1 *map_width + x1];
		return out;
	}
	else {
		float fq11 = mapping[y1 *map_width + x1];
		float fq21 = mapping[y1 *map_width + x2];
		float fq12 = mapping[y2 *map_width + x1];
		float fq22 = mapping[y2 *map_width + x2];

		if (-2 == fq11 || -2 == fq21 || -2 == fq12 || -2 == fq22)
		{
			return -2;
		}

		float out = fq11 * (x2 - x) * (y2 - y) + fq21 * (x - x1) * (y2 - y) + fq12 * (x2 - x) * (y - y1) + fq22 * (x - x1) * (y - y1);

		return out;
	}
	 

}


__device__ float mini_bilinear_interpolation(float x, float y, int map_width, float *mapping)
{
	//map_width = 129;

	//先找到这个点所对应的mini中的四个角点
	//然后将这四个点算出来
	//最后双线性插值

	int index_x1 = floor(x / 16);
	int index_y1 = floor((y-1301) / 16);
	int index_x2 = index_x1 + 1;
	int index_y2 = index_y1 + 1;

	int x1 = index_x1 * 16;
	int y1 = index_y1 * 16 + 1301;
	int x2 = x1 + 16;
	int y2 = y1 + 16;

	//因为我生成的表比原来大，所以无需考虑边界条件
	//fq_xy
	float fq11 = mapping[index_y1 *map_width + index_x1];
	float fq21 = mapping[index_y1 *map_width + index_x2];
	float fq12 = mapping[index_y2 *map_width + index_x1];
	float fq22 = mapping[index_y2 *map_width + index_x2];

	float out = (fq11 * (x2 - x) * (y2 - y) + fq21 * (x - x1) * (y2 - y) + fq12 * (x2 - x) * (y - y1) + fq22 * (x - x1) * (y - y1))/256.;

	return out;
}


__global__ void reconstruct_pointcloud_base_table(float * const xL_rotate_x,float * const xL_rotate_y,float * const single_pattern_mapping,float * const R_1,float * const confidence_map,
                                                        float * const phase_x,uint32_t img_height, uint32_t img_width,float b, float * const pointcloud,float * const depth)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

  
	const unsigned int serial_id = idy * img_width + idx;

	if (idx < img_width && idy < img_height)
	{
		/****************************************************************************/
		//phase to position
		float Xp = phase_x[serial_id] * d_dlp_width_ / (128.0*2*DF_PI); 
  
    	float Xcr = bilinear_interpolation(idx, idy,1920, xL_rotate_x);
        float Ycr = bilinear_interpolation(idx, idy, 1920,xL_rotate_y);
        float Xpr = bilinear_interpolation(Xp, (Ycr + 1) * 2000, 2000, single_pattern_mapping);
        float delta_X = std::abs(Xcr - Xpr);
		// float delta_X = Xpr -Xcr;
        float Z = b / delta_X;
	
		float X_L = Z * Xcr * R_1[0] + Z * Ycr * R_1[1] + Z * R_1[2];
		float Y_L = Z * Xcr * R_1[3] + Z * Ycr * R_1[4] + Z * R_1[5];
		float Z_L = Z * Xcr * R_1[6] + Z * Ycr * R_1[7] + Z * R_1[8];
 
  
		if(confidence_map[serial_id] > d_confidence_ && Z_L > 10 && Z_L< 60000 && Xp > 0)
		{
		    pointcloud[3 * serial_id + 0] = X_L;
		    pointcloud[3 * serial_id + 1] = Y_L;
		    pointcloud[3 * serial_id + 2] = Z_L; 
			
		    depth[serial_id] = Z_L; 
		}
		else
		{
		    pointcloud[3 * serial_id + 0] = 0;
		    pointcloud[3 * serial_id + 1] = 0;
		    pointcloud[3 * serial_id + 2] = 0; 
			
		    depth[serial_id] = 0; 
		}

		
		if (-2 == Xcr || -2 == Ycr || -2 == Xpr)
		{
			pointcloud[3 * serial_id + 0] = 0;
		    pointcloud[3 * serial_id + 1] = 0;
		    pointcloud[3 * serial_id + 2] = 0; 
			
		    depth[serial_id] = 0; 
		}
  
		/******************************************************************/


	}
}


__global__ void reconstruct_pointcloud_base_minitable(float* const xL_rotate_x, float* const xL_rotate_y, float* const single_pattern_minimapping, float* const R_1, float* const confidence_map,
	float* const phase_x, uint32_t img_height, uint32_t img_width, float b, float* const pointcloud, float* const depth)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;


	const unsigned int serial_id = idy * img_width + idx;

	if (idx < img_width && idy < img_height)
	{
		/****************************************************************************/
		//phase to position
		float Xp = phase_x[serial_id] * 1280.0 / (128.0 * 2 * DF_PI);

		float Xcr = bilinear_interpolation(idx, idy, 1920, xL_rotate_x);
		float Ycr = bilinear_interpolation(idx, idy, 1920, xL_rotate_y);
		//修改此处即可，需要自己写一个函数去查表
		float Xpr = mini_bilinear_interpolation(Xp, (Ycr + 1) * 2000, 128, single_pattern_minimapping);
		float delta_X = std::abs(Xcr - Xpr);
		float Z = b / delta_X;

		float X_L = Z * Xcr * R_1[0] + Z * Ycr * R_1[1] + Z * R_1[2];
		float Y_L = Z * Xcr * R_1[3] + Z * Ycr * R_1[4] + Z * R_1[5];
		float Z_L = Z * Xcr * R_1[6] + Z * Ycr * R_1[7] + Z * R_1[8];


		if (confidence_map[serial_id] > 10 && Z_L > 100 && Z_L < 2000)
		{
			pointcloud[3 * serial_id + 0] = X_L;
			pointcloud[3 * serial_id + 1] = Y_L;
			pointcloud[3 * serial_id + 2] = Z_L;

			depth[serial_id] = Z_L;
		}
		else
		{
			pointcloud[3 * serial_id + 0] = 0;
			pointcloud[3 * serial_id + 1] = 0;
			pointcloud[3 * serial_id + 2] = 0;

			depth[serial_id] = 0;
		}

		/******************************************************************/


	}
}


void reconstruct_copy_talbe_to_cuda_memory(float* mapping,float* rotate_x,float* rotate_y,float* r_1)
{
   
	CHECK(hipMemcpyAsync(d_R_1_, r_1, 3*3 * sizeof(float), hipMemcpyHostToDevice)); 
	CHECK(hipMemcpyAsync(d_single_pattern_mapping_, mapping, 4000*2000 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_xL_rotate_x_, rotate_x, image_height_*image_width_ * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_xL_rotate_y_, rotate_y, image_height_*image_width_ * sizeof(float), hipMemcpyHostToDevice));

}


void reconstruct_copy_minitalbe_to_cuda_memory(float* minimapping, float* rotate_x, float* rotate_y, float* r_1)
{

	CHECK(hipMemcpyAsync(d_R_1_, r_1, 3 * 3 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_single_pattern_minimapping_, minimapping, 128 * 128 * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_xL_rotate_x_, rotate_x, image_height_ * image_width_ * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpyAsync(d_xL_rotate_y_, rotate_y, image_height_ * image_width_ * sizeof(float), hipMemcpyHostToDevice));

}


void reconstruct_set_baseline(float b)
{
    d_baseline_ = b;  
}

void reconstruct_copy_pointcloud_from_cuda_memory(float* pointcloud)
{ 
	CHECK(hipMemcpy(pointcloud, d_point_cloud_map_, 3 * image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost));
}


void reconstruct_copy_confidence_from_cuda_memory(float* confidence)
{ 
	CHECK(hipMemcpy(confidence, d_confidence_list[3], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost)); 
}

void reconstruct_copy_depth_from_cuda_memory(float* depth)
{
	CHECK(hipMemcpy(depth, d_depth_map_, image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost)); 
} 

void reconstruct_copy_brightness_from_cuda_memory(unsigned char* brightness)
{
	CHECK(hipMemcpy(brightness, d_patterns_list[18], image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost)); 
}

  
void copy_phase_from_cuda_memory(float* phase_x,float* phase_y)
{
	CHECK(hipMemcpy(phase_x, d_unwrap_map_list[0], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost)); 
	CHECK(hipMemcpy(phase_y, d_unwrap_map_list[1], image_height_*image_width_ * sizeof(float), hipMemcpyDeviceToHost)); 
 
}
 
void copy_merge_brightness_from_cuda_memory(unsigned char* brightness)
{ 
	CHECK(hipMemcpy(brightness, d_brightness_, image_height_*image_width_ * sizeof(unsigned char), hipMemcpyDeviceToHost)); 
}

void reconstruct_cuda_malloc_memory()
{


	//为了防止重复的开辟内存，应先释放指针
	hipFree(d_single_pattern_mapping_);
    hipFree(d_xL_rotate_x_);
    hipFree(d_xL_rotate_y_);
    hipFree(d_R_1_);
	hipMalloc((void**)&d_single_pattern_mapping_, 4000*2000 * sizeof(float)); 
	hipMalloc((void**)&d_xL_rotate_x_, image_height_*image_width_ * sizeof(float)); 
	hipMalloc((void**)&d_xL_rotate_y_, image_height_*image_width_ * sizeof(float)); 
	hipMalloc((void**)&d_R_1_, 3*3 * sizeof(float)); 

  
}


void reconstruct_cuda_minimalloc_memory()
{
	
	//为了防止重复的开辟内存，应先释放指针
	hipFree(d_single_pattern_minimapping_);
    hipFree(d_xL_rotate_x_);
    hipFree(d_xL_rotate_y_);
    hipFree(d_R_1_);
	hipMalloc((void**)&d_single_pattern_minimapping_, 128*128 * sizeof(float)); 
	hipMalloc((void**)&d_xL_rotate_x_, image_height_*image_width_ * sizeof(float)); 
	hipMalloc((void**)&d_xL_rotate_y_, image_height_*image_width_ * sizeof(float)); 
	hipMalloc((void**)&d_R_1_, 3*3 * sizeof(float)); 

  
}

void reconstruct_cuda_free_memory()
{
    hipFree(d_single_pattern_mapping_);
    hipFree(d_xL_rotate_x_);
    hipFree(d_xL_rotate_y_);
    hipFree(d_R_1_);
 
}

void reconstruct_cuda_free_minimemory()
{
    hipFree(d_single_pattern_minimapping_);
    hipFree(d_xL_rotate_x_);
    hipFree(d_xL_rotate_y_);
    hipFree(d_R_1_);
 
}


 


/****************************************************************************************************/